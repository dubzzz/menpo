#include "hip/hip_runtime.h"
#include "HOG.h"
#include "CudaChecks.hpp"

#define MAX_THREADS_1D 256
#define MAX_THREADS_2D  16

void ZhuRamananHOGdescriptor_cpu(double *inputImage,
    int cellHeightAndWidthInPixels,
    unsigned int imageHeight,
    unsigned int imageWidth,
    unsigned int numberOfChannels,
    double *descriptorMatrix);
void DalalTriggsHOGdescriptor_cpu(double *inputImage,
    unsigned int numberOfOrientationBins,
    unsigned int cellHeightAndWidthInPixels,
    unsigned int blockHeightAndWidthInCells,
    bool signedOrUnsignedGradientsBool,
    double l2normClipping,
    unsigned int imageHeight,
    unsigned int imageWidth,
    unsigned int numberOfChannels,
    double *descriptorVector);

#ifdef __HIPCC__
    #define CUDA_MAJOR 2
    #define CUDA_MINOR 0
    int __DEVICE_COUNT__(-1); // -1 means to be done
    bool is_cuda_available();
    __device__ double atomicAdd(double *address, double val);
    
    // ZhuRamananHOG
    __global__ void kernel_image_ZhuRamananHOGdescriptor(double *d_hist,
        const dim3 visible,
        const dim3 blocks,
        const double *d_inputImage,
        const unsigned int imageHeight,
        const unsigned int imageWidth,
        const unsigned int numberOfChannels,
        const int cellHeightAndWidthInPixels);
    void ZhuRamananHOGdescriptor_cuda(double *inputImage,
        int cellHeightAndWidthInPixels,
        unsigned int imageHeight,
        unsigned int imageWidth,
        unsigned int numberOfChannels,
        double *descriptorMatrix);
    
    // DalalTriggsHOG
    __global__ void kernel_image_DalalTriggsHOGdescriptor(double *d_h,
        const dim3 h_dims,
        const double *d_inputImage,
        const unsigned int imageHeight,
        const unsigned int imageWidth,
        const unsigned int numberOfChannels,
        const unsigned int numberOfOrientationBins,
        const unsigned int cellHeightAndWidthInPixels,
        const unsigned signedOrUnsignedGradients,
        const double binsSize);
    void DalalTriggsHOGdescriptor_cuda(double *inputImage,
        unsigned int numberOfOrientationBins,
        unsigned int cellHeightAndWidthInPixels,
        unsigned int blockHeightAndWidthInCells,
        bool signedOrUnsignedGradientsBool,
        double l2normClipping,
        unsigned int imageHeight,
        unsigned int imageWidth,
        unsigned int numberOfChannels,
        double *descriptorVector);
#endif

/**
    HOG methods
*/

HOG::HOG(unsigned int windowHeight, unsigned int windowWidth,
         unsigned int numberOfChannels, unsigned int method,
         unsigned int numberOfOrientationBins,
         unsigned int cellHeightAndWidthInPixels,
         unsigned int blockHeightAndWidthInCells, bool enableSignedGradients,
         double l2normClipping) {
    unsigned int descriptorLengthPerBlock = 0,
                 numberOfBlocksPerWindowVertically = 0,
                 numberOfBlocksPerWindowHorizontally = 0;

    if (method == 1) {
        descriptorLengthPerBlock = blockHeightAndWidthInCells *
                                   blockHeightAndWidthInCells *
                                   numberOfOrientationBins;
        numberOfBlocksPerWindowVertically = 1 +
        (windowHeight - blockHeightAndWidthInCells*cellHeightAndWidthInPixels)
        / cellHeightAndWidthInPixels;
        numberOfBlocksPerWindowHorizontally = 1 +
        (windowWidth - blockHeightAndWidthInCells * cellHeightAndWidthInPixels)
        / cellHeightAndWidthInPixels;
    }
    else if (method==2) {
        descriptorLengthPerBlock = 27 + 4;
        numberOfBlocksPerWindowVertically =
        (unsigned int)round((double)windowHeight /
                            (double)cellHeightAndWidthInPixels) - 2;
        numberOfBlocksPerWindowHorizontally =
        (unsigned int)round((double)windowWidth /
                            (double)cellHeightAndWidthInPixels) - 2;
    }
    this->method = method;
    this->numberOfOrientationBins = numberOfOrientationBins;
    this->cellHeightAndWidthInPixels = cellHeightAndWidthInPixels;
    this->blockHeightAndWidthInCells = blockHeightAndWidthInCells;
    this->enableSignedGradients = enableSignedGradients;
    this->l2normClipping = l2normClipping;
    this->numberOfBlocksPerWindowHorizontally =
                    numberOfBlocksPerWindowHorizontally;
    this->numberOfBlocksPerWindowVertically =
                    numberOfBlocksPerWindowVertically;
    this->descriptorLengthPerBlock = descriptorLengthPerBlock;
    this->descriptorLengthPerWindow = numberOfBlocksPerWindowHorizontally *
                                      numberOfBlocksPerWindowVertically *
                                      descriptorLengthPerBlock;
    this->windowHeight = windowHeight;
    this->windowWidth = windowWidth;
    this->numberOfChannels = numberOfChannels;
}

HOG::~HOG() {}

void HOG::apply(double *windowImage, double *descriptorVector)
{
    #ifdef __HIPCC__
        if (is_cuda_available())
        {
            if (this->method == 1)
                DalalTriggsHOGdescriptor_cuda(windowImage, this->numberOfOrientationBins,
                    this->cellHeightAndWidthInPixels,
                    this->blockHeightAndWidthInCells,
                    this->enableSignedGradients,
                    this->l2normClipping, this->windowHeight,
                    this->windowWidth, this->numberOfChannels,
                    descriptorVector);
            else
                ZhuRamananHOGdescriptor_cuda(windowImage, this->cellHeightAndWidthInPixels,
                    this->windowHeight, this->windowWidth,
                    this->numberOfChannels, descriptorVector);
        }
        else
        {
            if (this->method == 1)
                DalalTriggsHOGdescriptor_cpu(windowImage, this->numberOfOrientationBins,
                    this->cellHeightAndWidthInPixels,
                    this->blockHeightAndWidthInCells,
                    this->enableSignedGradients,
                    this->l2normClipping, this->windowHeight,
                    this->windowWidth, this->numberOfChannels,
                    descriptorVector);
            else
                ZhuRamananHOGdescriptor_cpu(windowImage, this->cellHeightAndWidthInPixels,
                    this->windowHeight, this->windowWidth,
                    this->numberOfChannels, descriptorVector);
        }
    #else
        if (this->method == 1)
            DalalTriggsHOGdescriptor_cpu(windowImage, this->numberOfOrientationBins,
                this->cellHeightAndWidthInPixels,
                this->blockHeightAndWidthInCells,
                this->enableSignedGradients,
                this->l2normClipping, this->windowHeight,
                this->windowWidth, this->numberOfChannels,
                descriptorVector);
        else
            ZhuRamananHOGdescriptor_cpu(windowImage, this->cellHeightAndWidthInPixels,
                this->windowHeight, this->windowWidth,
                this->numberOfChannels, descriptorVector);
    #endif
}

/**
    CPU code
*/

// ZHU & RAMANAN: Face Detection, Pose Estimation and Landmark Localization
//                in the Wild
void ZhuRamananHOGdescriptor_cpu(double *inputImage,
                                 int cellHeightAndWidthInPixels,
                                 unsigned int imageHeight, unsigned int imageWidth,
                                 unsigned int numberOfChannels,
                                 double *descriptorMatrix) {
    // unit vectors used to compute gradient orientation
    double uu[9] = {1.0000, 0.9397, 0.7660, 0.500, 0.1736, -0.1736, -0.5000,
                    -0.7660, -0.9397};
    double vv[9] = {0.0000, 0.3420, 0.6428, 0.8660, 0.9848, 0.9848, 0.8660,
                    0.6428, 0.3420};
    int x, y, o;

    // memory for caching orientation histograms & their norms
    int blocks[2];
    blocks[0] = (int)round((double)imageHeight /
                           (double)cellHeightAndWidthInPixels);
    blocks[1] = (int)round((double)imageWidth /
                           (double)cellHeightAndWidthInPixels);
    double *hist = (double *)calloc(blocks[0] * blocks[1] * 18, sizeof(double));
    double *norm = (double *)calloc(blocks[0] * blocks[1], sizeof(double));

    // memory for HOG features
    int out[3];
    out[0] = max(blocks[0]-2, 0);
    out[1] = max(blocks[1]-2, 0);
    out[2] = 27+4;

    int visible[2];
    visible[0] = blocks[0] * cellHeightAndWidthInPixels;
    visible[1] = blocks[1] * cellHeightAndWidthInPixels;

    for (x = 1; x < visible[1] - 1; x++) {
        for (y = 1; y < visible[0] - 1; y++) {
            // compute gradient
            // first channel
            double *s = inputImage + min(x, imageWidth-2) * imageHeight +
                        min(y, imageHeight-2);
            double dy = *(s + 1) - *(s - 1);
            double dx = *(s + imageHeight) - *(s - imageHeight);
            double v = dx * dx + dy * dy;
            // rest of channels
            for (unsigned int z = 1; z < numberOfChannels; z++) {
                s += imageHeight * imageWidth;
                double dy2 = *(s + 1) - *(s - 1);
                double dx2 = *(s + imageHeight) - *(s - imageHeight);
                double v2 = dx2 * dx2 + dy2 * dy2;
                // pick channel with strongest gradient
                if (v2 > v) {
                    v = v2;
                    dx = dx2;
                    dy = dy2;
                }
            }

            // snap to one of 18 orientations
            double best_dot = 0;
            int best_o = 0;
            for (o = 0; o < 9; o++) {
                double dot = uu[o] * dx + vv[o] * dy;
                if (dot > best_dot) {
                    best_dot = dot;
                    best_o = o;
                }
                else if (-dot > best_dot) {
                    best_dot = - dot;
                    best_o = o + 9;
                }
            }

            // add to 4 histograms around pixel using linear interpolation
            double xp = ((double)x + 0.5) /
                        (double)cellHeightAndWidthInPixels - 0.5;
            double yp = ((double)y + 0.5) /
                        (double)cellHeightAndWidthInPixels - 0.5;
            int ixp = (int)floor(xp);
            int iyp = (int)floor(yp);
            double vx0 = xp - ixp;
            double vy0 = yp - iyp;
            double vx1 = 1.0 - vx0;
            double vy1 = 1.0 - vy0;
            v = sqrt(v);

            if (ixp >= 0 && iyp >= 0)
                *(hist + ixp*blocks[0] + iyp + best_o*blocks[0]*blocks[1])
                    += vx1 * vy1 * v;

            if (ixp+1 < blocks[1] && iyp >= 0)
                *(hist + (ixp+1)*blocks[0] + iyp + best_o*blocks[0]*blocks[1])
                    += vx0 * vy1 * v;

            if (ixp >= 0 && iyp+1 < blocks[0])
                *(hist + ixp*blocks[0] + (iyp+1) + best_o*blocks[0]*blocks[1])
                    += vx1 * vy0 * v;

            if (ixp+1 < blocks[1] && iyp+1 < blocks[0])
                *(hist + (ixp+1)*blocks[0] + (iyp+1) + best_o*blocks[0]*blocks[1])
                    += vx0 * vy0 * v;
        }
    }

    // compute energy in each block by summing over orientations
    for (int o = 0; o < 9; o++) {
        double *src1 = hist + o * blocks[0] * blocks[1];
        double *src2 = hist + (o + 9) * blocks[0] * blocks[1];
        double *dst = norm;
        double *end = norm + blocks[1] * blocks[0];
        while (dst < end) {
            *(dst++) += (*src1 + *src2) * (*src1 + *src2);
            src1++;
            src2++;
        }
    }

    // compute features
    for (x = 0; x < out[1]; x++) {
        for (y = 0; y < out[0]; y++) {
            double *dst = descriptorMatrix + x * out[0] + y;
            double *src, *p, n1, n2, n3, n4;

            p = norm + (x + 1) * blocks[0] + y + 1;
            n1 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks[0]) +
                            *(p + blocks[0] + 1) + eps);
            p = norm + (x + 1) * blocks[0] + y;
            n2 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks[0]) +
                            *(p + blocks[0] + 1) + eps);
            p = norm + x * blocks[0] + y + 1;
            n3 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks[0]) +
                            *(p + blocks[0] + 1) + eps);
            p = norm + x * blocks[0] + y;
            n4 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks[0]) +
                            *(p + blocks[0] + 1) + eps);

            double t1 = 0;
            double t2 = 0;
            double t3 = 0;
            double t4 = 0;

            // contrast-sensitive features
            src = hist + (x + 1) * blocks[0] + (y + 1);
            for (int o = 0; o < 18; o++) {
                double h1 = min(*src * n1, 0.2);
                double h2 = min(*src * n2, 0.2);
                double h3 = min(*src * n3, 0.2);
                double h4 = min(*src * n4, 0.2);
                *dst = 0.5 * (h1 + h2 + h3 + h4);
                t1 += h1;
                t2 += h2;
                t3 += h3;
                t4 += h4;
                dst += out[0] * out[1];
                src += blocks[0] * blocks[1];
            }

            // contrast-insensitive features
            src = hist + (x + 1) * blocks[0] + (y + 1);
            for (int o = 0; o < 9; o++) {
                double sum = *src + *(src + 9 * blocks[0] * blocks[1]);
                double h1 = min(sum * n1, 0.2);
                double h2 = min(sum * n2, 0.2);
                double h3 = min(sum * n3, 0.2);
                double h4 = min(sum * n4, 0.2);
                *dst = 0.5 * (h1 + h2 + h3 + h4);
                dst += out[0] * out[1];
                src += blocks[0] * blocks[1];
            }

            // texture features
            *dst = 0.2357 * t1;
            dst += out[0] * out[1];
            *dst = 0.2357 * t2;
            dst += out[0] * out[1];
            *dst = 0.2357 * t3;
            dst += out[0] * out[1];
            *dst = 0.2357 * t4;
        }
    }
    free(hist);
    free(norm);
}


// DALAL & TRIGGS: Histograms of Oriented Gradients for Human Detection
void DalalTriggsHOGdescriptor_cpu(double *inputImage,
                                  unsigned int numberOfOrientationBins,
                                  unsigned int cellHeightAndWidthInPixels,
                                  unsigned int blockHeightAndWidthInCells,
                                  bool signedOrUnsignedGradientsBool,
                                  double l2normClipping, unsigned int imageHeight,
                                  unsigned int imageWidth,
                                  unsigned int numberOfChannels,
                                  double *descriptorVector) {
    
    numberOfOrientationBins = (int)numberOfOrientationBins;
    cellHeightAndWidthInPixels = (double)cellHeightAndWidthInPixels;
    blockHeightAndWidthInCells = (int)blockHeightAndWidthInCells;

    unsigned int signedOrUnsignedGradients;
    
    if (signedOrUnsignedGradientsBool) {
        signedOrUnsignedGradients = 1;
    } else {
        signedOrUnsignedGradients = 0;
    }

    int hist1 = 2 + (imageHeight / cellHeightAndWidthInPixels);
    int hist2 = 2 + (imageWidth / cellHeightAndWidthInPixels);

    double binsSize = (1 + (signedOrUnsignedGradients == 1)) *
                      pi / numberOfOrientationBins;

    float *dx = new float[numberOfChannels];
    float *dy = new float[numberOfChannels];
    float gradientOrientation, gradientMagnitude, tempMagnitude, 
          Xc, Yc, Oc, blockNorm;
    int x1 = 0, x2 = 0, y1 = 0, y2 = 0, bin1 = 0, descriptorIndex = 0;
    unsigned int x, y, i, j, k, bin2;

    vector<vector<vector<double> > > h(hist1, vector<vector<double> >
                                       (hist2, vector<double>
                                        (numberOfOrientationBins, 0.0 ) ) );
    vector<vector<vector<double> > > block(blockHeightAndWidthInCells, vector<vector<double> >
                                           (blockHeightAndWidthInCells, vector<double>
                                            (numberOfOrientationBins, 0.0) ) );

    //Calculate gradients (zero padding)
    for(unsigned int y = 0; y < imageHeight; y++) {
        for(unsigned int x = 0; x < imageWidth; x++) {
            if (x == 0) {
                for (unsigned int z = 0; z < numberOfChannels; z++)
                    dx[z] = inputImage[y + (x + 1) * imageHeight +
                                       z * imageHeight * imageWidth];
            }
            else {
                if (x == imageWidth - 1) {
                    for (unsigned int z = 0; z < numberOfChannels; z++)
                        dx[z] = -inputImage[y + (x - 1) * imageHeight +
                                            z * imageHeight * imageWidth];
                }
                else {
                    for (unsigned int z = 0; z < numberOfChannels; z++)
                        dx[z] = inputImage[y + (x + 1) * imageHeight +
                                           z * imageHeight * imageWidth] -
                                inputImage[y + (x - 1) * imageHeight +
                                           z * imageHeight * imageWidth];
                }
            }

            if(y == 0) {
                for (unsigned int z = 0; z < numberOfChannels; z++)
                    dy[z] = -inputImage[y + 1 + x * imageHeight +
                                        z * imageHeight * imageWidth];
            }
            else {
                if (y == imageHeight - 1) {
                    for (unsigned int z = 0; z < numberOfChannels; z++)
                        dy[z] = inputImage[y - 1 + x * imageHeight +
                                           z * imageHeight * imageWidth];
                }
                else {
                    for (unsigned int z = 0; z < numberOfChannels; z++)
                        dy[z] = -inputImage[y + 1 + x * imageHeight +
                                            z * imageHeight * imageWidth] +
                                 inputImage[y - 1 + x * imageHeight +
                                            z * imageHeight * imageWidth];
                }
            }

            // choose dominant channel based on magnitude
            gradientMagnitude = sqrt(dx[0] * dx[0] + dy[0] * dy[0]);
            gradientOrientation= atan2(dy[0], dx[0]);
            if (numberOfChannels > 1) {
                tempMagnitude = gradientMagnitude;
                for (unsigned int cli = 1; cli < numberOfChannels; ++cli) {
                    tempMagnitude= sqrt(dx[cli] * dx[cli] + dy[cli] * dy[cli]);
                    if (tempMagnitude > gradientMagnitude) {
                        gradientMagnitude = tempMagnitude;
                        gradientOrientation = atan2(dy[cli], dx[cli]);
                    }
                }
            }

            if (gradientOrientation < 0)
                gradientOrientation += pi +
                                       (signedOrUnsignedGradients == 1) * pi;

            // trilinear interpolation
            bin1 = (gradientOrientation / binsSize) - 1;
            bin2 = bin1 + 1;
            x1   = x / cellHeightAndWidthInPixels;
            x2   = x1 + 1;
            y1   = y / cellHeightAndWidthInPixels;
            y2   = y1 + 1;

            Xc = (x1 + 1 - 1.5) * cellHeightAndWidthInPixels + 0.5;
            Yc = (y1 + 1 - 1.5) * cellHeightAndWidthInPixels + 0.5;
            Oc = (bin1 + 1 + 1 - 1.5) * binsSize;

            if (bin2 == numberOfOrientationBins)
                bin2 = 0;

            if (bin1 < 0)
                bin1 = numberOfOrientationBins - 1;

            h[y1][x1][bin1] = h[y1][x1][bin1] + gradientMagnitude *
                              (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                              (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                              (1-((gradientOrientation-Oc)/binsSize));
            h[y1][x1][bin2] = h[y1][x1][bin2] + gradientMagnitude *
                              (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                              (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                              (((gradientOrientation-Oc)/binsSize));
            h[y2][x1][bin1] = h[y2][x1][bin1] + gradientMagnitude *
                              (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                              (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                              (1-((gradientOrientation-Oc)/binsSize));
            h[y2][x1][bin2] = h[y2][x1][bin2] + gradientMagnitude *
                              (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                              (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                              (((gradientOrientation-Oc)/binsSize));
            h[y1][x2][bin1] = h[y1][x2][bin1] + gradientMagnitude *
                              (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                              (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                              (1-((gradientOrientation-Oc)/binsSize));
            h[y1][x2][bin2] = h[y1][x2][bin2] + gradientMagnitude *
                              (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                              (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                              (((gradientOrientation-Oc)/binsSize));
            h[y2][x2][bin1] = h[y2][x2][bin1] + gradientMagnitude *
                              (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                              (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                              (1-((gradientOrientation-Oc)/binsSize));
            h[y2][x2][bin2] = h[y2][x2][bin2] + gradientMagnitude *
                              (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                              (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                              (((gradientOrientation-Oc)/binsSize));
        }
    }

    //Block normalization
    for(x = 1; x < hist2 - blockHeightAndWidthInCells; x++) {
        for (y = 1; y < hist1 - blockHeightAndWidthInCells; y++) {
            blockNorm = 0;
            for (i = 0; i < blockHeightAndWidthInCells; i++)
                for(j = 0; j < blockHeightAndWidthInCells; j++)
                    for(k = 0; k < numberOfOrientationBins; k++)
                        blockNorm += h[y+i][x+j][k] * h[y+i][x+j][k];

            blockNorm = sqrt(blockNorm);
            for (i = 0; i < blockHeightAndWidthInCells; i++) {
                for(j = 0; j < blockHeightAndWidthInCells; j++) {
                    for(k = 0; k < numberOfOrientationBins; k++) {
                        if (blockNorm > 0) {
                            block[i][j][k] = h[y+i][x+j][k] / blockNorm;
                            if (block[i][j][k] > l2normClipping)
                                block[i][j][k] = l2normClipping;
                        }
                    }
                }
            }

            blockNorm = 0;
            for (i = 0; i < blockHeightAndWidthInCells; i++)
                for(j = 0; j < blockHeightAndWidthInCells; j++)
                    for(k = 0; k < numberOfOrientationBins; k++)
                        blockNorm += block[i][j][k] * block[i][j][k];

            blockNorm = sqrt(blockNorm);
            for (i = 0; i < blockHeightAndWidthInCells; i++) {
                for(j = 0; j < blockHeightAndWidthInCells; j++) {
                    for(k = 0; k < numberOfOrientationBins; k++) {
                        if (blockNorm > 0)
                            descriptorVector[descriptorIndex] =
                                block[i][j][k] / blockNorm;
                        else
                            descriptorVector[descriptorIndex] = 0.0;
                        descriptorIndex++;
                    }
                }
            }
        }
    }
    delete[] dx;
    delete[] dy;
}

/**
    CUDA/GPU code
*/

#ifdef __HIPCC__

/**
    Check if CUDA is available or not
    and compatible or not
    The check itself is only done one time.
    Result is then stored in the variable __DEVICE_COUNT__
    
    from https://github.com/dubzzz/cuda-compile-4-cuda-and-noncuda/
*/
bool is_cuda_available()
{
    if (__DEVICE_COUNT__ == -1)
    {
        int deviceCount;
        hipError_t e = hipGetDeviceCount(&deviceCount);
        if (e != hipSuccess)
            __DEVICE_COUNT__ = 0;
        else
        {
            __DEVICE_COUNT__ = 0;
            
            // for each GPU check if it has the required compute capability
            for (int i(0) ; i != deviceCount ; i++)
            {
                hipDeviceProp_t prop;
                e = hipGetDeviceProperties(&prop, i);
                // check compute capability
                if (e == hipSuccess && (prop.major > CUDA_MAJOR || (prop.major == CUDA_MAJOR && prop.minor >= CUDA_MINOR)))
                {
                    hipSetDevice(i);
                    __DEVICE_COUNT__++;
                }
            }
        }
    }
    return __DEVICE_COUNT__ != 0;
}

__device__ double atomicAdd(double* address, double val) // http://stackoverflow.com/questions/16882253/cuda-atomicadd-produces-wrong-result
{
    unsigned long long int* address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;
    do
    {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    }
    while (assumed != old);
    return __longlong_as_double(old);
}

// ZHU & RAMANAN: Face Detection, Pose Estimation and Landmark Localization
//                in the Wild
__global__ void kernel_image_ZhuRamananHOGdescriptor(double *d_hist,
                                                     const dim3 visible,
                                                     const dim3 blocks,
                                                     const double *d_inputImage,
                                                     const unsigned int imageHeight,
                                                     const unsigned int imageWidth,
                                                     const unsigned int numberOfChannels,
                                                     const int cellHeightAndWidthInPixels)
{
    // Retrieve pixel position
    int x = blockIdx.x * blockDim.x + threadIdx.x +1;
    int y = blockIdx.y * blockDim.y + threadIdx.y +1;
     
    // Check if position is inside the image and not on borders
    // full check: if (x < 1 || y < 1 || x >= visible.x -1 || y >= visible.y -1), but x>=1 and y>=1
    if (x >= visible.x -1 || y >= visible.y -1)
        return;
    
    // Usefull variables
    // unit vectors used to compute gradient orientation
    double uu[9] = {1.0000, 0.9397, 0.7660, 0.500, 0.1736, -0.1736, -0.5000, -0.7660, -0.9397};
    double vv[9] = {0.0000, 0.3420, 0.6428, 0.8660, 0.9848, 0.9848, 0.8660, 0.6428, 0.3420};
    
    // compute gradient
    // first channel
    const double *s = d_inputImage + min(x, imageWidth-2) * imageHeight + min(y, imageHeight-2);
    double dy = *(s + 1) - *(s - 1);
    double dx = *(s + imageHeight) - *(s - imageHeight);
    double v = dx * dx + dy * dy;
    // rest of channels
    for (unsigned int z = 1; z < numberOfChannels; z++)
    {
        s += imageHeight * imageWidth;
        double dy2 = *(s + 1) - *(s - 1);
        double dx2 = *(s + imageHeight) - *(s - imageHeight);
        double v2 = dx2 * dx2 + dy2 * dy2;
        // pick channel with strongest gradient
        if (v2 > v) {
            v = v2;
            dx = dx2;
            dy = dy2;
        }
    }

    // snap to one of 18 orientations
    double best_dot = 0;
    int best_o = 0;
    for (int o = 0; o < 9; o++)
    {
        double dot = uu[o] * dx + vv[o] * dy;
        if (dot > best_dot)
        {
            best_dot = dot;
            best_o = o;
        }
        else if (-dot > best_dot)
        {
            best_dot = - dot;
            best_o = o + 9;
        }
    }

    // add to 4 histograms around pixel using linear interpolation
    double xp = ((double) x + 0.5) / (double) cellHeightAndWidthInPixels - 0.5;
    double yp = ((double) y + 0.5) / (double) cellHeightAndWidthInPixels - 0.5;
    int ixp = (int) floor(xp);
    int iyp = (int) floor(yp);
    double vx0 = xp - ixp;
    double vy0 = yp - iyp;
    double vx1 = 1.0 - vx0;
    double vy1 = 1.0 - vy0;
    v = sqrt(v);

    if (ixp >= 0 && iyp >= 0)
        atomicAdd(&d_hist[ixp*blocks.x + iyp + best_o*blocks.x*blocks.y], vx1 * vy1 * v);

    if (ixp+1 < blocks.y && iyp >= 0)
        atomicAdd(&d_hist[(ixp+1)*blocks.x + iyp + best_o*blocks.x*blocks.y], vx0 * vy1 * v);

    if (ixp >= 0 && iyp+1 < blocks.x)
        atomicAdd(&d_hist[ixp*blocks.x + (iyp+1) + best_o*blocks.x*blocks.y], vx1 * vy0 * v);

    if (ixp+1 < blocks.y && iyp+1 < blocks.x)
        atomicAdd(&d_hist[(ixp+1)*blocks.x + (iyp+1) + best_o*blocks.x*blocks.y], vx0 * vy0 * v);
}

void ZhuRamananHOGdescriptor_cuda(double *inputImage,
                                  int cellHeightAndWidthInPixels,
                                  unsigned int imageHeight, unsigned int imageWidth,
                                  unsigned int numberOfChannels,
                                  double *descriptorMatrix)
{
    // Compute histograms
    // memory for caching orientation histograms
    
    dim3 blocks((int) round((double) imageHeight / (double) cellHeightAndWidthInPixels), (int) round((double) imageWidth / (double) cellHeightAndWidthInPixels), 0);
    dim3 visible(blocks.x * cellHeightAndWidthInPixels, blocks.y * cellHeightAndWidthInPixels, 0);
    double *d_hist;
    cudaErrorCheck(hipMalloc(&d_hist, blocks.x * blocks.y * 18 * sizeof(double)));
    cudaErrorCheck(hipMemset(d_hist, 0., blocks.x * blocks.y * 18 * sizeof(double)));
    
    double *d_inputImage;
    cudaErrorCheck(hipMalloc(&d_inputImage, imageHeight * imageWidth * sizeof(double)));
    cudaErrorCheck(hipMemcpy(d_inputImage, inputImage, imageHeight * imageWidth * sizeof(double), hipMemcpyHostToDevice));
    
    const dim3 dimBlock(MAX_THREADS_2D, MAX_THREADS_2D, 1);
    const dim3 dimGrid((visible.x -2 + dimBlock.x -1)/dimBlock.x, (visible.y -2 + dimBlock.y -1)/dimBlock.y, 1); // x in [1,visible.x -1] ; y in [1,visible.y -1]
    kernel_image_ZhuRamananHOGdescriptor<<<dimGrid, dimBlock>>>(d_hist, visible, blocks,
                                                                d_inputImage, imageHeight, imageWidth, numberOfChannels,
                                                                cellHeightAndWidthInPixels);
    cudaErrorCheck(hipFree(d_inputImage));
    
    // memory for caching orientation histograms on cpu
    double hist[blocks.x * blocks.y * 18];
    cudaErrorCheck(hipMemcpy(hist, d_hist, blocks.x * blocks.y * 18 * sizeof(double), hipMemcpyDeviceToHost));
    cudaErrorCheck(hipFree(d_hist));
    
    // memory for caching orientation histograms norms
    double *norm = (double *)calloc(blocks.x * blocks.y, sizeof(double));
    
    // memory for HOG features
    int out[3];
    out[0] = max(blocks.x-2, 0);
    out[1] = max(blocks.y-2, 0);
    out[2] = 27+4;

    // compute energy in each block by summing over orientations
    
    for (int o = 0; o < 9; o++) {
        double *src1 = hist + o * blocks.x * blocks.y;
        double *src2 = hist + (o + 9) * blocks.x * blocks.y;
        double *dst = norm;
        double *end = norm + blocks.y * blocks.x;
        while (dst < end) {
            *(dst++) += (*src1 + *src2) * (*src1 + *src2);
            src1++;
            src2++;
        }
    }

    // compute features
    
    for (int x = 0; x < out[1]; x++) {
        for (int y = 0; y < out[0]; y++) {
            double *dst = descriptorMatrix + x * out[0] + y;
            double *src, *p, n1, n2, n3, n4;

            p = norm + (x + 1) * blocks.x + y + 1;
            n1 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks.x) +
                            *(p + blocks.x + 1) + eps);
            p = norm + (x + 1) * blocks.x + y;
            n2 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks.x) +
                            *(p + blocks.x + 1) + eps);
            p = norm + x * blocks.x + y + 1;
            n3 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks.x) +
                            *(p + blocks.x + 1) + eps);
            p = norm + x * blocks.x + y;
            n4 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks.x) +
                            *(p + blocks.x + 1) + eps);

            double t1 = 0;
            double t2 = 0;
            double t3 = 0;
            double t4 = 0;

            // contrast-sensitive features
            src = hist + (x + 1) * blocks.x + (y + 1);
            for (int o = 0; o < 18; o++) {
                double h1 = min(*src * n1, 0.2);
                double h2 = min(*src * n2, 0.2);
                double h3 = min(*src * n3, 0.2);
                double h4 = min(*src * n4, 0.2);
                *dst = 0.5 * (h1 + h2 + h3 + h4);
                t1 += h1;
                t2 += h2;
                t3 += h3;
                t4 += h4;
                dst += out[0] * out[1];
                src += blocks.x * blocks.y;
            }

            // contrast-insensitive features
            src = hist + (x + 1) * blocks.x + (y + 1);
            for (int o = 0; o < 9; o++) {
                double sum = *src + *(src + 9 * blocks.x * blocks.y);
                double h1 = min(sum * n1, 0.2);
                double h2 = min(sum * n2, 0.2);
                double h3 = min(sum * n3, 0.2);
                double h4 = min(sum * n4, 0.2);
                *dst = 0.5 * (h1 + h2 + h3 + h4);
                dst += out[0] * out[1];
                src += blocks.x * blocks.y;
            }

            // texture features
            *dst = 0.2357 * t1;
            dst += out[0] * out[1];
            *dst = 0.2357 * t2;
            dst += out[0] * out[1];
            *dst = 0.2357 * t3;
            dst += out[0] * out[1];
            *dst = 0.2357 * t4;
        }
    }
    free(norm);
}

__global__ void kernel_image_DalalTriggsHOGdescriptor(double *d_h,
                                                      const dim3 h_dims,
                                                      const double *d_inputImage,
                                                      const unsigned int imageHeight,
                                                      const unsigned int imageWidth,
                                                      const unsigned int numberOfChannels,
                                                      const unsigned int numberOfOrientationBins,
                                                      const unsigned int cellHeightAndWidthInPixels,
                                                      const unsigned signedOrUnsignedGradients,
                                                      const double binsSize)
{
    // Retrieve pixel position
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int factor_z_dim = h_dims.x * h_dims.y;
    unsigned int factor_y_dim = h_dims.x;
     
    // Check if position is inside the image
    if (x >= imageWidth || y >= imageHeight)
        return;
    
    // Compute deltas
    double dx[3], dy[3];
    
    if (x == 0)
    {
        for (unsigned int z = 0; z < numberOfChannels; z++)
            dx[z] = d_inputImage[y + (x + 1) * imageHeight + z * imageHeight * imageWidth];
    }
    else
    {
        if (x == imageWidth - 1)
        {
            for (unsigned int z = 0; z < numberOfChannels; z++)
                dx[z] = -d_inputImage[y + (x - 1) * imageHeight + z * imageHeight * imageWidth];
        }
        else
        {
            for (unsigned int z = 0; z < numberOfChannels; z++)
                dx[z] = d_inputImage[y + (x + 1) * imageHeight + z * imageHeight * imageWidth] - d_inputImage[y + (x - 1) * imageHeight + z * imageHeight * imageWidth];
        }
    }

    if(y == 0)
    {
        for (unsigned int z = 0; z < numberOfChannels; z++)
            dy[z] = -d_inputImage[y + 1 + x * imageHeight + z * imageHeight * imageWidth];
    }
    else
    {
        if (y == imageHeight - 1)
        {
            for (unsigned int z = 0; z < numberOfChannels; z++)
                dy[z] = d_inputImage[y - 1 + x * imageHeight + z * imageHeight * imageWidth];
        }
        else
        {
            for (unsigned int z = 0; z < numberOfChannels; z++)
                dy[z] = -d_inputImage[y + 1 + x * imageHeight + z * imageHeight * imageWidth] + d_inputImage[y - 1 + x * imageHeight + z * imageHeight * imageWidth];
        }
    }

    // Choose dominant channel based on magnitude
    double gradientMagnitude = sqrt(dx[0] * dx[0] + dy[0] * dy[0]);
    double gradientOrientation = atan2(dy[0], dx[0]);
    if (numberOfChannels > 1)
    {
        double tempMagnitude = gradientMagnitude;
        for (unsigned int cli = 1 ; cli < numberOfChannels ; ++cli)
        {
            tempMagnitude= sqrt(dx[cli] * dx[cli] + dy[cli] * dy[cli]);
            if (tempMagnitude > gradientMagnitude)
            {
                gradientMagnitude = tempMagnitude;
                gradientOrientation = atan2(dy[cli], dx[cli]);
            }
        }
    }

    if (gradientOrientation < 0)
        gradientOrientation += pi + (signedOrUnsignedGradients == 1) * pi;

    // Trilinear interpolation
    int bin1 = (gradientOrientation / binsSize) - 1;
    unsigned int bin2 = bin1 + 1;
    int x1   = x / cellHeightAndWidthInPixels;
    int x2   = x1 + 1;
    int y1   = y / cellHeightAndWidthInPixels;
    int y2   = y1 + 1;
    
    double Xc = (x1 + 1 - 1.5) * cellHeightAndWidthInPixels + 0.5;
    double Yc = (y1 + 1 - 1.5) * cellHeightAndWidthInPixels + 0.5;
    double Oc = (bin1 + 1 + 1 - 1.5) * binsSize;
    
    if (bin2 == numberOfOrientationBins)
        bin2 = 0;
    
    if (bin1 < 0)
        bin1 = numberOfOrientationBins - 1;
    
    atomicAdd(&d_h[y1 + x1 * factor_y_dim + bin1 * factor_z_dim], gradientMagnitude *
                                                                  (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                                                                  (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                                                                  (1-((gradientOrientation-Oc)/binsSize)));
    atomicAdd(&d_h[y1 + x1 * factor_y_dim + bin2 * factor_z_dim], gradientMagnitude *
                                                                  (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                                                                  (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                                                                  (((gradientOrientation-Oc)/binsSize)));
    atomicAdd(&d_h[y2 + x1 * factor_y_dim + bin1 * factor_z_dim], gradientMagnitude *
                                                                  (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                                                                  (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                                                                  (1-((gradientOrientation-Oc)/binsSize)));
    atomicAdd(&d_h[y2 + x1 * factor_y_dim + bin2 * factor_z_dim], gradientMagnitude *
                                                                  (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                                                                  (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                                                                  (((gradientOrientation-Oc)/binsSize)));
    atomicAdd(&d_h[y1 + x2 * factor_y_dim + bin1 * factor_z_dim], gradientMagnitude *
                                                                  (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                                                                  (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                                                                  (1-((gradientOrientation-Oc)/binsSize)));
    atomicAdd(&d_h[y1 + x2 * factor_y_dim + bin2 * factor_z_dim], gradientMagnitude *
                                                                  (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                                                                  (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                                                                  (((gradientOrientation-Oc)/binsSize)));
    atomicAdd(&d_h[y2 + x2 * factor_y_dim + bin1 * factor_z_dim], gradientMagnitude *
                                                                  (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                                                                  (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                                                                  (1-((gradientOrientation-Oc)/binsSize)));
    atomicAdd(&d_h[y2 + x2 * factor_y_dim + bin2 * factor_z_dim], gradientMagnitude *
                                                                  (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                                                                  (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                                                                  (((gradientOrientation-Oc)/binsSize)));
}

// DALAL & TRIGGS: Histograms of Oriented Gradients for Human Detection
void DalalTriggsHOGdescriptor_cuda(double *inputImage,
                                   unsigned int numberOfOrientationBins,
                                   unsigned int cellHeightAndWidthInPixels,
                                   unsigned int blockHeightAndWidthInCells,
                                   bool signedOrUnsignedGradientsBool,
                                   double l2normClipping, unsigned int imageHeight,
                                   unsigned int imageWidth,
                                   unsigned int numberOfChannels,
                                   double *descriptorVector) {
    
    //Calculate gradients (zero padding)
    //using CUDA
    
    const int hist1 = 2 + (imageHeight / cellHeightAndWidthInPixels);
    const int hist2 = 2 + (imageWidth / cellHeightAndWidthInPixels);
    const dim3 h_dims(hist1, hist2, numberOfOrientationBins);
    const unsigned int factor_z_dim = h_dims.x * h_dims.y;
    const unsigned int factor_y_dim = h_dims.x;
    double *d_h;
    cudaErrorCheck(hipMalloc(&d_h, h_dims.x * h_dims.y * h_dims.z * sizeof(double)));
    cudaErrorCheck(hipMemset(d_h, 0., h_dims.x * h_dims.y * h_dims.z * sizeof(double)));
    
    double *d_inputImage;
    cudaErrorCheck(hipMalloc(&d_inputImage, imageHeight * imageWidth * sizeof(double)));
    cudaErrorCheck(hipMemcpy(d_inputImage, inputImage, imageHeight * imageWidth * sizeof(double), hipMemcpyHostToDevice));
    
    const dim3 dimBlock(MAX_THREADS_2D, MAX_THREADS_2D, 1);
    const dim3 dimGrid((imageWidth + dimBlock.x -1)/dimBlock.x, (imageHeight + dimBlock.y -1)/dimBlock.y, 1);
    kernel_image_DalalTriggsHOGdescriptor<<<dimGrid, dimBlock>>>(d_h, h_dims,
                                                                 d_inputImage, imageHeight, imageWidth, numberOfChannels,
                                                                 numberOfOrientationBins, cellHeightAndWidthInPixels,
                                                                 signedOrUnsignedGradientsBool ? 1 : 0 /*signedOrUnsignedGradients*/,
                                                                 (1 + (signedOrUnsignedGradientsBool ? 1 : 0)) * pi / numberOfOrientationBins /*binsSize*/);
    hipDeviceSynchronize(); // block until the device is finished
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
        cudaErrorRaise(error);
    
    double h[h_dims.x * h_dims.y * h_dims.z];
    cudaErrorCheck(hipMemcpy(h, d_h, h_dims.x * h_dims.y * h_dims.z * sizeof(double), hipMemcpyDeviceToHost));
    
    cudaErrorCheck(hipFree(d_h));
    cudaErrorCheck(hipFree(d_inputImage));
    
    //Block normalization
    
    int descriptorIndex(0);
    vector<vector<vector<double> > > block(blockHeightAndWidthInCells, vector<vector<double> >
                                           (blockHeightAndWidthInCells, vector<double>
                                            (numberOfOrientationBins, 0.0) ) );
    
    for (unsigned int x = 1; x < hist2 - blockHeightAndWidthInCells; x++) {
        for (unsigned int y = 1; y < hist1 - blockHeightAndWidthInCells; y++) {
            float blockNorm(0);
            for (unsigned int i = 0; i < blockHeightAndWidthInCells; i++)
                for (unsigned int j = 0; j < blockHeightAndWidthInCells; j++)
                    for (unsigned int k = 0; k < numberOfOrientationBins; k++)
                        blockNorm += h[y+i + (x+j) * factor_y_dim + k * factor_z_dim] * h[y+i + (x+j) * factor_y_dim + k * factor_z_dim];

            blockNorm = sqrt(blockNorm);
            for (unsigned int i = 0; i < blockHeightAndWidthInCells; i++) {
                for (unsigned int j = 0; j < blockHeightAndWidthInCells; j++) {
                    for (unsigned int k = 0; k < numberOfOrientationBins; k++) {
                        if (blockNorm > 0) {
                            block[i][j][k] = h[y+i + (x+j) * factor_y_dim + k * factor_z_dim] / blockNorm;
                            if (block[i][j][k] > l2normClipping)
                                block[i][j][k] = l2normClipping;
                        }
                    }
                }
            }

            blockNorm = 0;
            for (unsigned int i = 0; i < blockHeightAndWidthInCells; i++)
                for (unsigned int j = 0; j < blockHeightAndWidthInCells; j++)
                    for (unsigned int k = 0; k < numberOfOrientationBins; k++)
                        blockNorm += block[i][j][k] * block[i][j][k];

            blockNorm = sqrt(blockNorm);
            for (unsigned int i = 0; i < blockHeightAndWidthInCells; i++) {
                for (unsigned int j = 0; j < blockHeightAndWidthInCells; j++) {
                    for (unsigned int k = 0; k < numberOfOrientationBins; k++, descriptorIndex++) {
                        if (blockNorm > 0)
                            descriptorVector[descriptorIndex] = block[i][j][k] / blockNorm;
                        else
                            descriptorVector[descriptorIndex] = 0.0;
                    }
                }
            }
        }
    }
}

#endif

