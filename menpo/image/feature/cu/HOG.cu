#include "hip/hip_runtime.h"
#include "HOG.h"
#include "check_cuda_errors.hpp"
#include "Python.h"

#define MAX_THREADS_1D      256
#define MAX_THREADS_1D_SMALL 64
#define MAX_THREADS_2D       16

#define MAX_THREADS_3DX       4
#define MAX_THREADS_3DY       4
#define MAX_THREADS_3DZ      16

HOG::HOG(unsigned int windowHeight, unsigned int windowWidth,
         unsigned int numberOfChannels, unsigned int method,
         unsigned int numberOfOrientationBins,
         unsigned int cellHeightAndWidthInPixels,
         unsigned int blockHeightAndWidthInCells, bool enableSignedGradients,
         double l2normClipping) {
    unsigned int descriptorLengthPerBlock = 0,
                 numberOfBlocksPerWindowVertically = 0,
                 numberOfBlocksPerWindowHorizontally = 0;

    if (method == 1) {
        descriptorLengthPerBlock = blockHeightAndWidthInCells *
                                   blockHeightAndWidthInCells *
                                   numberOfOrientationBins;
        numberOfBlocksPerWindowVertically = 1 +
        (windowHeight - blockHeightAndWidthInCells*cellHeightAndWidthInPixels)
        / cellHeightAndWidthInPixels;
        numberOfBlocksPerWindowHorizontally = 1 +
        (windowWidth - blockHeightAndWidthInCells * cellHeightAndWidthInPixels)
        / cellHeightAndWidthInPixels;
    }
    else if (method==2) {
        descriptorLengthPerBlock = 27 + 4;
        numberOfBlocksPerWindowVertically =
        (unsigned int)round((double)windowHeight /
                            (double)cellHeightAndWidthInPixels) - 2;
        numberOfBlocksPerWindowHorizontally =
        (unsigned int)round((double)windowWidth /
                            (double)cellHeightAndWidthInPixels) - 2;
    }
    this->method = method;
    this->numberOfOrientationBins = numberOfOrientationBins;
    this->cellHeightAndWidthInPixels = cellHeightAndWidthInPixels;
    this->blockHeightAndWidthInCells = blockHeightAndWidthInCells;
    this->enableSignedGradients = enableSignedGradients;
    this->l2normClipping = l2normClipping;
    this->numberOfBlocksPerWindowHorizontally =
                    numberOfBlocksPerWindowHorizontally;
    this->numberOfBlocksPerWindowVertically =
                    numberOfBlocksPerWindowVertically;
    this->descriptorLengthPerBlock = descriptorLengthPerBlock;
    this->descriptorLengthPerWindow = numberOfBlocksPerWindowHorizontally *
                                      numberOfBlocksPerWindowVertically *
                                      descriptorLengthPerBlock;
    this->windowHeight = windowHeight;
    this->windowWidth = windowWidth;
    this->numberOfChannels = numberOfChannels;
}

HOG::~HOG() {
}


void HOG::applyOnChunk(double *windowImage, double *descriptorVector) {
    if (this->method == 1)
        PyErr_SetString(PyExc_RuntimeError,
                        "HOG::applyOnChunk is not implemented for DalalTriggs");
    else
        ZhuRamananHOGdescriptor(windowImage, this->cellHeightAndWidthInPixels,
                                this->windowHeight, this->windowWidth,
                                this->numberOfChannels, descriptorVector);
}

void HOG::applyOnImage(const ImageWindowIterator &iwi, const double *image,
                       double *outputImage, int *windowsCenters) {
    __CLOG__
    double *d_image = 0;
    if (this->method == 1) {
        const unsigned int imageHeight = iwi._imageHeight;
        const unsigned int imageWidth = iwi._imageWidth;
        const unsigned int numberOfChannels = iwi._numberOfChannels;
        
        __START__
        cudaErrorCheck_goto(hipMalloc(&d_image, imageHeight * imageWidth * numberOfChannels * sizeof(double)));
        cudaErrorCheck_goto(hipMemcpy(d_image, image, imageHeight * imageWidth * numberOfChannels * sizeof(double), hipMemcpyHostToDevice));
        __STOP("@ Malloc & Memcpy for <image> @")
        this->DalalTriggsHOGdescriptorOnImage(iwi, d_image, outputImage, windowsCenters);
        __START__
        cudaErrorCheck_goto(hipFree(d_image));
        d_image = 0;
        __STOP("@ Free for <image> @")
    } else
        PyErr_SetString(PyExc_RuntimeError,
                        "HOG::applyOnImage is not implemented for ZhuRamanan");
    return;

onfailure:
    hipFree(d_image);
    return;
}

bool HOG::isApplyOnImage() {
    if (this->method == 1) // easier to read this way
        return true;
    else
        return false;
}

// ZHU & RAMANAN: Face Detection, Pose Estimation and Landmark Localization
//                in the Wild
void ZhuRamananHOGdescriptor(double *inputImage,
                             int cellHeightAndWidthInPixels,
                             unsigned int imageHeight, unsigned int imageWidth,
                             unsigned int numberOfChannels,
                             double *descriptorMatrix) {
    // unit vectors used to compute gradient orientation
    double uu[9] = {1.0000, 0.9397, 0.7660, 0.500, 0.1736, -0.1736, -0.5000,
                    -0.7660, -0.9397};
    double vv[9] = {0.0000, 0.3420, 0.6428, 0.8660, 0.9848, 0.9848, 0.8660,
                    0.6428, 0.3420};
    int x, y, o;

    // memory for caching orientation histograms & their norms
    int blocks[2];
    blocks[0] = (int)round((double)imageHeight /
                           (double)cellHeightAndWidthInPixels);
    blocks[1] = (int)round((double)imageWidth /
                           (double)cellHeightAndWidthInPixels);
    double *hist = (double *)calloc(blocks[0] * blocks[1] * 18, sizeof(double));
    double *norm = (double *)calloc(blocks[0] * blocks[1], sizeof(double));

    // memory for HOG features
    int out[3];
    out[0] = max(blocks[0]-2, 0);
    out[1] = max(blocks[1]-2, 0);
    out[2] = 27+4;

    int visible[2];
    visible[0] = blocks[0] * cellHeightAndWidthInPixels;
    visible[1] = blocks[1] * cellHeightAndWidthInPixels;

    for (x = 1; x < visible[1] - 1; x++) {
        for (y = 1; y < visible[0] - 1; y++) {
            // compute gradient
            // first channel
            double *s = inputImage + min(x, imageWidth-2) * imageHeight +
                        min(y, imageHeight-2);
            double dy = *(s + 1) - *(s - 1);
            double dx = *(s + imageHeight) - *(s - imageHeight);
            double v = dx * dx + dy * dy;
            // rest of channels
            for (unsigned int z = 1; z < numberOfChannels; z++) {
                s += imageHeight * imageWidth;
                double dy2 = *(s + 1) - *(s - 1);
                double dx2 = *(s + imageHeight) - *(s - imageHeight);
                double v2 = dx2 * dx2 + dy2 * dy2;
                // pick channel with strongest gradient
                if (v2 > v) {
                    v = v2;
                    dx = dx2;
                    dy = dy2;
                }
            }

            // snap to one of 18 orientations
            double best_dot = 0;
            int best_o = 0;
            for (o = 0; o < 9; o++) {
                double dot = uu[o] * dx + vv[o] * dy;
                if (dot > best_dot) {
                    best_dot = dot;
                    best_o = o;
                }
                else if (-dot > best_dot) {
                    best_dot = - dot;
                    best_o = o + 9;
                }
            }

            // add to 4 histograms around pixel using linear interpolation
            double xp = ((double)x + 0.5) /
                        (double)cellHeightAndWidthInPixels - 0.5;
            double yp = ((double)y + 0.5) /
                        (double)cellHeightAndWidthInPixels - 0.5;
            int ixp = (int)floor(xp);
            int iyp = (int)floor(yp);
            double vx0 = xp - ixp;
            double vy0 = yp - iyp;
            double vx1 = 1.0 - vx0;
            double vy1 = 1.0 - vy0;
            v = sqrt(v);

            if (ixp >= 0 && iyp >= 0)
                *(hist + ixp*blocks[0] + iyp + best_o*blocks[0]*blocks[1])
                    += vx1 * vy1 * v;

            if (ixp+1 < blocks[1] && iyp >= 0)
                *(hist + (ixp+1)*blocks[0] + iyp + best_o*blocks[0]*blocks[1])
                    += vx0 * vy1 * v;

            if (ixp >= 0 && iyp+1 < blocks[0])
                *(hist + ixp*blocks[0] + (iyp+1) + best_o*blocks[0]*blocks[1])
                    += vx1 * vy0 * v;

            if (ixp+1 < blocks[1] && iyp+1 < blocks[0])
                *(hist + (ixp+1)*blocks[0] + (iyp+1) + best_o*blocks[0]*blocks[1])
                    += vx0 * vy0 * v;
        }
    }

    // compute energy in each block by summing over orientations
    for (int o = 0; o < 9; o++) {
        double *src1 = hist + o * blocks[0] * blocks[1];
        double *src2 = hist + (o + 9) * blocks[0] * blocks[1];
        double *dst = norm;
        double *end = norm + blocks[1] * blocks[0];
        while (dst < end) {
            *(dst++) += (*src1 + *src2) * (*src1 + *src2);
            src1++;
            src2++;
        }
    }

    // compute features
    for (x = 0; x < out[1]; x++) {
        for (y = 0; y < out[0]; y++) {
            double *dst = descriptorMatrix + x * out[0] + y;
            double *src, *p, n1, n2, n3, n4;

            p = norm + (x + 1) * blocks[0] + y + 1;
            n1 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks[0]) +
                            *(p + blocks[0] + 1) + eps);
            p = norm + (x + 1) * blocks[0] + y;
            n2 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks[0]) +
                            *(p + blocks[0] + 1) + eps);
            p = norm + x * blocks[0] + y + 1;
            n3 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks[0]) +
                            *(p + blocks[0] + 1) + eps);
            p = norm + x * blocks[0] + y;
            n4 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks[0]) +
                            *(p + blocks[0] + 1) + eps);

            double t1 = 0;
            double t2 = 0;
            double t3 = 0;
            double t4 = 0;

            // contrast-sensitive features
            src = hist + (x + 1) * blocks[0] + (y + 1);
            for (int o = 0; o < 18; o++) {
                double h1 = min(*src * n1, 0.2);
                double h2 = min(*src * n2, 0.2);
                double h3 = min(*src * n3, 0.2);
                double h4 = min(*src * n4, 0.2);
                *dst = 0.5 * (h1 + h2 + h3 + h4);
                t1 += h1;
                t2 += h2;
                t3 += h3;
                t4 += h4;
                dst += out[0] * out[1];
                src += blocks[0] * blocks[1];
            }

            // contrast-insensitive features
            src = hist + (x + 1) * blocks[0] + (y + 1);
            for (int o = 0; o < 9; o++) {
                double sum = *src + *(src + 9 * blocks[0] * blocks[1]);
                double h1 = min(sum * n1, 0.2);
                double h2 = min(sum * n2, 0.2);
                double h3 = min(sum * n3, 0.2);
                double h4 = min(sum * n4, 0.2);
                *dst = 0.5 * (h1 + h2 + h3 + h4);
                dst += out[0] * out[1];
                src += blocks[0] * blocks[1];
            }

            // texture features
            *dst = 0.2357 * t1;
            dst += out[0] * out[1];
            *dst = 0.2357 * t2;
            dst += out[0] * out[1];
            *dst = 0.2357 * t3;
            dst += out[0] * out[1];
            *dst = 0.2357 * t4;
        }
    }
    free(hist);
    free(norm);
}

void HOG::DalalTriggsHOGdescriptorOnImage(const ImageWindowIterator &iwi,
                                          double *d_image,
                                          double *outputImage,
                                          int *windowsCenters) {
    __CLOG__
    int rowCenter, columnCenter;
    
    // Define useful variables
    
    const unsigned int numWindows = iwi._numberOfWindowsVertically*iwi._numberOfWindowsHorizontally;
    const int hist1 = 2 + (this->windowHeight / this->cellHeightAndWidthInPixels);
    const int hist2 = 2 + (this->windowWidth / this->cellHeightAndWidthInPixels);
    
    double *d_blockNorm = 0, *d_block = 0, *d_outputImage = 0;
    
    // Each block has to compute its blockNorm[x][y]
    // stored into d_blockNorm[x-1 + blockNorm_dims.x*(y-1)]
    // Computation done with reduce-pattern
    //   x: unsigned int x = 1; x < hist2 - blockHeightAndWidthInCells; x++
    //      => requires (hist2 - blockHeightAndWidthInCells -1) Blocks
    //   y: unsigned int y = 1; y < hist1 - blockHeightAndWidthInCells; y++
    //      => requires (hist1 - blockHeightAndWidthInCells -1) Blocks
    // Each block works on 3D-Threads
    //   i: unsigned int i = 0; i < blockHeightAndWidthInCells; i++
    //   j: unsigned int j = 0; j < blockHeightAndWidthInCells; j++
    //      => usually: blockHeightAndWidthInCells=2
    //   k: unsigned int k = 0; k < numberOfOrientationBins; k++
    //      => usually: larger than blockHeightAndWidthInCells
    const dim3 blockNorm_dims(hist2 - blockHeightAndWidthInCells -1,
                              hist1 - blockHeightAndWidthInCells -1);
    const dim3 dimBlock_norm(MAX_THREADS_3DX, MAX_THREADS_3DY, MAX_THREADS_3DZ);
    const dim3 dimGrid_norm(blockNorm_dims.x*iwi._numberOfWindowsHorizontally,
                            blockNorm_dims.y*iwi._numberOfWindowsVertically, 1);
    
    // Each thread has to compute a single value of d_block for a given window
    // - block[i,j,k,x,y]
    //
    // Idea of the size of block vector for a given window:
    //    0 <= i < 2 (=blockHeightAndWidthInCells)
    //    0 <= j < 2 (=blockHeightAndWidthInCells)
    //    0 <= k < 9 (=numberOfOrientationBins)
    //    0 <= x < 1 (=blockNorm_dims.x)
    //    0 <= y < 1 (=blockNorm_dims.y)
    // = 2x2x9 = 36 elements to compute << 256 (=MAX_THREADS_1D)
    //(= block_size)
    //
    // For some configuration, it might be greater than MAX_THREADS_1D,
    // the kernel handles this case
    //
    // A kernel's block corresponds to a window
    // each kernel's block has to compute its elements of d_block
    const dim3 dimBlock_block(MAX_THREADS_1D_SMALL, 1, 1);
    const dim3 dimGrid_block(iwi._numberOfWindowsHorizontally,
                             iwi._numberOfWindowsVertically, 1);
    const unsigned int block_size = blockHeightAndWidthInCells
                                    * blockHeightAndWidthInCells
                                    * numberOfOrientationBins
                                    * blockNorm_dims.x
                                    * blockNorm_dims.y;
    
    // Each thread has to compute one element of outputImage
    // The kernel block is equivalent to a window, each block computes its own
    // "descriptorVector" - which is immediately written into outputImage 
    const dim3 dimBlock_desc(dimBlock_block);
    const dim3 dimGrid_desc(dimGrid_block);
    
    const unsigned long long int d_outputImage_size_t = iwi._numberOfWindowsVertically
            * iwi._numberOfWindowsHorizontally
            * this->descriptorLengthPerWindow * sizeof(double);
    
    const dim3 h_dims(hist1, hist2, this->numberOfOrientationBins);
    const unsigned long int h_size = h_dims.x * h_dims.y * h_dims.z * numWindows;
    const unsigned long long int d_h_size_t = h_size * sizeof(double);
    double *d_h = 0; // contains all the histograms
    
    const dim3 dimBlock(MAX_THREADS_2D, MAX_THREADS_2D, 1);
    const dim3 dimGrid((this->windowWidth * iwi._numberOfWindowsHorizontally + dimBlock.x -1)/dimBlock.x, (this->windowHeight * iwi._numberOfWindowsVertically + dimBlock.y -1)/dimBlock.y, 1);
    
    
    // Pre-allocate CUDA memory for DalalTriggsHOGdescriptor
    //
    // Allocating/Deleting memory takes lots of time for small vectors
    // Allocating/Deleting vectors before remove the cost of this operation
    
    __START__
    cudaErrorCheck_goto(hipMalloc(&d_blockNorm, blockNorm_dims.x
                                                 * blockNorm_dims.y
                                                 * numWindows
                                                 * sizeof(double)));
    cudaErrorCheck_goto(hipMalloc(&d_block, cellHeightAndWidthInPixels
                                             * cellHeightAndWidthInPixels
                                             * numberOfOrientationBins
                                             * blockNorm_dims.x
                                             * blockNorm_dims.y
                                             * numWindows
                                             * sizeof(double)));
    
    // Compute all the histograms together using CUDA
    //   h_dims: dimension of one histogram
    //   numWindows: number of histograms to compute
    //
    //       +--+--+  +--+
    // d_h = |h0|h1|..|hn|
    //       +--+--+  +--+
    //
    // where hx is a histogram
    //   and n = numWindows
    
    cudaErrorCheck_goto(hipMalloc(&d_h, d_h_size_t));
    cudaErrorCheck_goto(hipMemset(d_h, 0., d_h_size_t));
    
    // Allocate memory for the CUDA version of outputImage
    
    cudaErrorCheck_goto(hipMalloc(&d_outputImage, d_outputImage_size_t));
    __STOP("@ Malloc & Memset for <output, d_h..> @")
    
    // Compute values for histograms
    __START__
    DalalTriggsHOGdescriptor_compute_histograms<<<dimGrid, dimBlock>>>(d_h, h_dims,
                                                                       d_image, iwi._imageHeight, iwi._imageWidth,
                                                                       this->windowHeight, this->windowWidth, this->numberOfChannels,
                                                                       this->numberOfOrientationBins, this->cellHeightAndWidthInPixels,
                                                                       this->enableSignedGradients ? 1 : 0 /*signedOrUnsignedGradients*/,
                                                                       (1 + (this->enableSignedGradients ? 1 : 0)) * pi / this->numberOfOrientationBins /*binsSize*/,
                                                                       numWindows, iwi._numberOfWindowsVertically,
                                                                       iwi._numberOfWindowsHorizontally,
                                                                       iwi._enablePadding, iwi._windowStepVertical, iwi._windowStepHorizontal);
    cudaErrorCheck_goto(hipDeviceSynchronize()); // block until the device is finished
    __STOP("@ Kernel: compute_histograms @")
    
    // Histogram normalization
    // Evaluate blockNorm based on d_h
    __START__
    cudaErrorCheck_goto(hipMemset(d_blockNorm, 0., blockNorm_dims.x * blockNorm_dims.y * numWindows * sizeof(double)));
    DalalTriggsHOGdescriptor_compute_blocknorm<<<dimGrid_norm,
                                                 dimBlock_norm,
                                                 MAX_THREADS_3DX
                                                 * MAX_THREADS_3DY
                                                 * MAX_THREADS_3DZ
                                                 * sizeof(double)>>>
                                                    (d_blockNorm, blockNorm_dims,
                                                     d_h, h_dims,
                                                     numberOfOrientationBins,
                                                     blockHeightAndWidthInCells,
                                                     iwi._numberOfWindowsVertically);
    cudaErrorCheck_goto(hipDeviceSynchronize()); // block until the device is finished
    __STOP("@ Kernel: compute_blocknorm @")
    
    // Compute block
    __START__
    DalalTriggsHOGdescriptor_compute_block<<<dimGrid_block,
                                             dimBlock_block>>>
                                                (d_block,
                                                 d_blockNorm, blockNorm_dims,
                                                 d_h, h_dims,
                                                 numberOfOrientationBins,
                                                 blockHeightAndWidthInCells,
                                                 l2normClipping,
                                                 iwi._numberOfWindowsVertically,
                                                 block_size);
    cudaErrorCheck_goto(hipDeviceSynchronize()); // block until the device is finished
    __STOP("@ Kernel: compute_block @")
    
    // Evaluate blockNorm based on d_block
    __START__
    DalalTriggsHOGdescriptor_compute_blocknorm2<<<dimGrid_norm,
                                                  dimBlock_norm,
                                                  MAX_THREADS_3DX
                                                  * MAX_THREADS_3DY
                                                  * MAX_THREADS_3DZ
                                                  * sizeof(double)>>>
                                                    (d_blockNorm, blockNorm_dims,
                                                     d_block,
                                                     numberOfOrientationBins,
                                                     blockHeightAndWidthInCells,
                                                     iwi._numberOfWindowsVertically);
    cudaErrorCheck_goto(hipDeviceSynchronize()); // block until the device is finished
    __STOP("@ Kernel: compute_blocknorm2 @")
    
    // Compute outputImage
    __START__
    DalalTriggsHOGdescriptor_compute_outputImage<<<dimGrid_desc,
                                                   dimBlock_desc>>>
                                                    (d_outputImage,
                                                     d_block,
                                                     d_blockNorm,
                                                     blockNorm_dims,
                                                     numberOfOrientationBins,
                                                     blockHeightAndWidthInCells,
                                                     numWindows,
                                                     iwi._numberOfWindowsVertically,
                                                     block_size);
    cudaErrorCheck_goto(hipDeviceSynchronize()); // block until the device is finished
    __STOP("@ Kernel: compute_outputImage @")
    
    // Histogram normalization
    // & windowsCenters initialization
    //
    // Everything is done with native-C code (ie. without any CUDA implementation)
    __START__
    for (unsigned int windowIndexVertical = 0; windowIndexVertical < iwi._numberOfWindowsVertically; windowIndexVertical++) {
        for (unsigned int windowIndexHorizontal = 0; windowIndexHorizontal < iwi._numberOfWindowsHorizontally; windowIndexHorizontal++) {
            // Find window limits
            if (!iwi._enablePadding) {
                rowCenter = windowIndexVertical*iwi._windowStepVertical
                            + (int)round((double)iwi._windowHeight / 2.0) - 1;
                columnCenter = windowIndexHorizontal*iwi._windowStepHorizontal
                            + (int)round((double)iwi._windowWidth / 2.0) - 1;
            } else {
                rowCenter = windowIndexVertical*iwi._windowStepVertical;
                columnCenter = windowIndexHorizontal*iwi._windowStepHorizontal;
            }
            
            // Store results
            windowsCenters[windowIndexVertical+iwi._numberOfWindowsVertically*windowIndexHorizontal] = rowCenter;
            windowsCenters[windowIndexVertical+iwi._numberOfWindowsVertically*(windowIndexHorizontal+iwi._numberOfWindowsHorizontally)] = columnCenter;
        }
    }
    __STOP("@ Histogram Normalization @")
    
    __START__
    cudaErrorCheck_goto(hipMemcpy(
            outputImage, d_outputImage,
            d_outputImage_size_t, hipMemcpyDeviceToHost));
    cudaErrorCheck_goto(hipFree(d_outputImage));
    d_outputImage = 0;
    
    cudaErrorCheck_goto(hipFree(d_h));
    d_h = 0;
    cudaErrorCheck_goto(hipFree(d_block));
    d_block = 0;
    cudaErrorCheck_goto(hipFree(d_blockNorm));
    d_blockNorm = 0;
    __STOP("@ Memcpy & Free for <output, d_h..> @")
    return;
    
onfailure:
    hipFree(d_h);
    hipFree(d_outputImage);
    hipFree(d_block);
    hipFree(d_blockNorm);
    return;
}

/* Kernels */

__device__ double atomicAdd(double* address, double val) {
    // http://stackoverflow.com/questions/16882253/cuda-atomicadd-produces-wrong-result
    unsigned long long int* address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;
    do
    {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

#define getInImage(i,j,k) ((i+rowFrom<0 || i+rowFrom>imageHeight-1 || j+columnFrom<0 || j+columnFrom>imageWidth-1) ? 0. : d_inputImage[(i+rowFrom) + imageHeight*((j+columnFrom) + imageWidth*k)])
__global__ void DalalTriggsHOGdescriptor_compute_histograms(double *d_h,
                                                               const dim3 h_dims,
                                                               const double *d_inputImage,
                                                               const unsigned int imageHeight,
                                                               const unsigned int imageWidth,
                                                               const unsigned int windowHeight,
                                                               const unsigned int windowWidth,
                                                               const unsigned int numberOfChannels,
                                                               const unsigned int numberOfOrientationBins,
                                                               const unsigned int cellHeightAndWidthInPixels,
                                                               const unsigned signedOrUnsignedGradients,
                                                               const double binsSize,
                                                               const int numHistograms,
                                                               const int numberOfWindowsVertically,
                                                               const int numberOfWindowsHorizontally,
                                                               const bool enablePadding,
                                                               const int windowStepVertical, const int windowStepHorizontal) {
    // Compute histograms values
    
    // Retrieve pixel position
    int x_ = blockIdx.x * blockDim.x + threadIdx.x;
    if (x_ >= numberOfWindowsHorizontally * windowWidth)
        return;
    int y_ = blockIdx.y * blockDim.y + threadIdx.y;
    if (y_ >= numberOfWindowsVertically * windowHeight)
        return;
    
    int x = x_ % windowWidth;
    int windowIndexHorizontal = x_ / windowWidth;
    
    int y = y_ % windowHeight;
    int windowIndexVertical = y_ / windowHeight;
    
    unsigned int factor_y_dim = h_dims.x;
    unsigned int factor_z_dim = factor_y_dim * h_dims.y;
    unsigned int factor_o_dim = factor_z_dim * h_dims.z;
    
    int offsetWindow = factor_o_dim * (windowIndexVertical + numberOfWindowsVertically * windowIndexHorizontal);
    int rowFrom, columnFrom;
    if (!enablePadding) {
        rowFrom = windowIndexVertical*windowStepVertical;
        columnFrom = windowIndexHorizontal*windowStepHorizontal;
    } else {
        rowFrom = windowIndexVertical*windowStepVertical - (int)round((double)windowHeight / 2.0) + 1;
        columnFrom = windowIndexHorizontal*windowStepHorizontal - (int)ceil((double)windowWidth / 2.0) + 1;
    }
     
    // Compute deltas
    double dx[3], dy[3];
    
    if (x == 0) {
        for (unsigned int z = 0; z < numberOfChannels; z++)
            dx[z] = getInImage(y, x+1, z);
    } else {
        if (x == windowWidth - 1) {
            for (unsigned int z = 0; z < numberOfChannels; z++)
                dx[z] = -getInImage(y, x-1, z);
        } else {
            for (unsigned int z = 0; z < numberOfChannels; z++)
                dx[z] = getInImage(y, x+1, z) - getInImage(y, x-1, z);
        }
    }

    if(y == 0) {
        for (unsigned int z = 0; z < numberOfChannels; z++)
            dy[z] = -getInImage(y+1, x, z);
    } else {
        if (y == windowHeight - 1) {
            for (unsigned int z = 0; z < numberOfChannels; z++)
                dy[z] = getInImage(y-1, x, z);
        } else {
            for (unsigned int z = 0; z < numberOfChannels; z++)
                dy[z] = -getInImage(y+1, x, z) + getInImage(y-1, x, z);
        }
    }

    // Choose dominant channel based on magnitude
    double gradientMagnitude = sqrt(dx[0] * dx[0] + dy[0] * dy[0]);
    double gradientOrientation = atan2(dy[0], dx[0]);
    if (numberOfChannels > 1) {
        double tempMagnitude = gradientMagnitude;
        for (unsigned int cli = 1 ; cli < numberOfChannels ; ++cli) {
            tempMagnitude= sqrt(dx[cli] * dx[cli] + dy[cli] * dy[cli]);
            if (tempMagnitude > gradientMagnitude) {
                gradientMagnitude = tempMagnitude;
                gradientOrientation = atan2(dy[cli], dx[cli]);
            }
        }
    }

    if (gradientOrientation < 0)
        gradientOrientation += pi + (signedOrUnsignedGradients == 1) * pi;

    // Trilinear interpolation
    int bin1 = (gradientOrientation / binsSize) - 1;
    unsigned int bin2 = bin1 + 1;
    int x1   = x / cellHeightAndWidthInPixels;
    int x2   = x1 + 1;
    int y1   = y / cellHeightAndWidthInPixels;
    int y2   = y1 + 1;
    
    double Xc = (x1 + 1 - 1.5) * cellHeightAndWidthInPixels + 0.5;
    double Yc = (y1 + 1 - 1.5) * cellHeightAndWidthInPixels + 0.5;
    double Oc = (bin1 + 1 + 1 - 1.5) * binsSize;
    
    if (bin2 == numberOfOrientationBins)
        bin2 = 0;
    
    if (bin1 < 0)
        bin1 = numberOfOrientationBins - 1;
    
    // Compute histograms
    //  using reduce-pattern
    //
    // d_h needs to be set to 0.
    
    atomicAdd(
            &d_h[offsetWindow + y1 + x1*factor_y_dim + bin1*factor_z_dim],
            gradientMagnitude *
                (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                (1-((gradientOrientation-Oc)/binsSize)));
    atomicAdd(
            &d_h[offsetWindow + y1 + x1*factor_y_dim + bin2*factor_z_dim],
            gradientMagnitude *
                (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                (((gradientOrientation-Oc)/binsSize)));
    atomicAdd(
            &d_h[offsetWindow + y2 + x1*factor_y_dim + bin1*factor_z_dim],
            gradientMagnitude *
                (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                (1-((gradientOrientation-Oc)/binsSize)));
    atomicAdd(
            &d_h[offsetWindow + y2 + x1*factor_y_dim + bin2*factor_z_dim],
            gradientMagnitude *
                (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                (((gradientOrientation-Oc)/binsSize)));
    atomicAdd(
            &d_h[offsetWindow + y1 + x2*factor_y_dim + bin1*factor_z_dim],
            gradientMagnitude *
                (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                (1-((gradientOrientation-Oc)/binsSize)));
    atomicAdd(
            &d_h[offsetWindow + y1 + x2*factor_y_dim + bin2*factor_z_dim],
            gradientMagnitude *
                (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                (((gradientOrientation-Oc)/binsSize)));
    atomicAdd(
            &d_h[offsetWindow + y2 + x2*factor_y_dim + bin1*factor_z_dim],
            gradientMagnitude *
                (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                (1-((gradientOrientation-Oc)/binsSize)));
    atomicAdd(
            &d_h[offsetWindow + y2 + x2*factor_y_dim + bin2*factor_z_dim],
            gradientMagnitude *
                (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                (((gradientOrientation-Oc)/binsSize)));
}

__global__ void DalalTriggsHOGdescriptor_compute_blocknorm(double *d_blockNorm,
                                                           const dim3 blockNorm_dims,
                                                           const double *d_h,
                                                           const dim3 h_dims,
                                                           const unsigned int numberOfOrientationBins,
                                                           const unsigned int blockHeightAndWidthInCells,
                                                           const unsigned int numberOfWindowsVertically) {
    // 2D-reduce to compute d_blockNorm for every (x,y)
    
    // Size of shared memory must be blockDim.x*blockDim.y*blockDim.z
    // and a power of 2
    extern __shared__ double cache[];
    
    // Compute factors
    unsigned int factor_z_dim = h_dims.x * h_dims.y;
    unsigned int factor_y_dim = h_dims.x;
    
    // Retrieve indice of the element
    unsigned int x = blockIdx.x;
    unsigned int y = blockIdx.y;
    unsigned int z = (y/blockNorm_dims.y) + numberOfWindowsVertically * (x/blockNorm_dims.x);
    x = (x % blockNorm_dims.x) +1;
    y = (y % blockNorm_dims.y) +1;
    unsigned int offsetH = h_dims.x * h_dims.y * h_dims.z * z;
    
    unsigned int i = threadIdx.x;
    unsigned int j = threadIdx.y;
    unsigned int k = threadIdx.z;
    unsigned int current_id = i + j*blockDim.x
                              + k*blockDim.x*blockDim.y;
    
    cache[current_id] = 0.;
    for (unsigned int i_=i ; i_ < blockHeightAndWidthInCells ; i_+=blockDim.x)
        for (unsigned int j_=j ; j_ < blockHeightAndWidthInCells ; j_+=blockDim.y)
            for (unsigned int k_=k ; k_ < numberOfOrientationBins ; k_+=blockDim.z)
                cache[current_id] += d_h[y+i_ + (x+j_) * factor_y_dim + k_ * factor_z_dim + offsetH]
                                     * d_h[y+i_ + (x+j_) * factor_y_dim + k_ * factor_z_dim + offsetH];
    __syncthreads();
    
    // Reduce operation
    // all threads in the current block have to compute d_blockNorm[x + hist2*y]
    int padding = blockDim.x*blockDim.y*blockDim.z/2;
    while (padding != 0) {
        if (current_id < padding)
            cache[current_id] += cache[current_id + padding];
        __syncthreads();
        padding /= 2;
    }
    
    if (i == 0 && j == 0 && k == 0)
        d_blockNorm[x-1 + blockNorm_dims.x*(y-1)
                    + blockNorm_dims.x*blockNorm_dims.y*z] = cache[0];
}

__global__ void DalalTriggsHOGdescriptor_compute_block(double *d_block,
                                                       const double *d_blockNorm,
                                                       const dim3 blockNorm_dims,
                                                       const double *d_h,
                                                       const dim3 h_dims,
                                                       const unsigned int numberOfOrientationBins,
                                                       const unsigned int blockHeightAndWidthInCells,
                                                       const double l2normClipping,
                                                       const unsigned int numberOfWindowsVertically,
                                                       const unsigned int block_size) {
    // Each thread has to compute one value of block[i,j,k,x,y]
    // for a given windows (blockIdx.x, blockIdx.y)
    
    // Compute window's index
    //unsigned int windowIndexHorizontal = blockIdx.x;
    //unsigned int windowIndexVertical = blockIdx.y;
    unsigned windowIndex = (blockIdx.y + numberOfWindowsVertically * blockIdx.x);
    
    // Compute factors
    unsigned int factor_z_dim = h_dims.x * h_dims.y;
    unsigned int factor_y_dim = h_dims.x;
    
    // Retrieve ids of elements to compute during this thread
    // In most of the cases, the loop should be called only once
    for (unsigned int elementIndex(threadIdx.x) ; elementIndex < block_size ; elementIndex += blockDim.x) {
        //elementIndex = i + j*blockHeightAndWidthInCells
        //               + k*blockHeightAndWidthInCells*blockHeightAndWidthInCells
        //               + (x-1)*blockHeightAndWidthInCells*blockHeightAndWidthInCells
        //                 *numberOfOrientationBins
        //               + (y-1)*blockHeightAndWidthInCells*blockHeightAndWidthInCells
        //                 *numberOfOrientationBins*blockNorm_dims.x
        unsigned int i = elementIndex;
        unsigned int j = i / blockHeightAndWidthInCells;
        i %= blockHeightAndWidthInCells;
        unsigned int k = j / blockHeightAndWidthInCells;
        j %= blockHeightAndWidthInCells;
        unsigned int x = k / numberOfOrientationBins;
        k %= numberOfOrientationBins;
        unsigned int y = (x / blockNorm_dims.x) +1;
        x = (x % blockNorm_dims.x) +1;
        
        unsigned int current_id = elementIndex
                                  + windowIndex * blockHeightAndWidthInCells
                                    * blockHeightAndWidthInCells
                                    * numberOfOrientationBins
                                    * blockNorm_dims.x * blockNorm_dims.y;
        
        double blockNorm = sqrt(d_blockNorm[x-1 + blockNorm_dims.x*(y-1)
                                            + blockNorm_dims.x*blockNorm_dims.y*windowIndex]);
        if (blockNorm > 0) {
            unsigned int offsetH = h_dims.x * h_dims.y * h_dims.z * windowIndex;
            double tmpValue = d_h[y+i + (x+j) * factor_y_dim
                                  + k * factor_z_dim + offsetH] / blockNorm;
            if (tmpValue > l2normClipping)
                d_block[current_id] = l2normClipping;
            else
                d_block[current_id] = tmpValue;
        } else
            d_block[current_id] = 0.;
    }
}

__global__ void DalalTriggsHOGdescriptor_compute_blocknorm2(double *d_blockNorm,
                                                            const dim3 blockNorm_dims,
                                                            const double *d_block,
                                                            const unsigned int numberOfOrientationBins,
                                                            const unsigned int blockHeightAndWidthInCells,
                                                            const unsigned int numberOfWindowsVertically) {
    // 2D-reduce to compute d_blockNorm for every (x,y)
    
    // Size of shared memory must be blockDim.x*blockDim.y*blockDim.z
    // and a power of 2
    extern __shared__ double cache[];
    
    // Retrieve indice of the element
    unsigned int x = blockIdx.x;
    unsigned int y = blockIdx.y;
    unsigned int z = (y/blockNorm_dims.y) + numberOfWindowsVertically * (x/blockNorm_dims.x);
    x = (x % blockNorm_dims.x) +1;
    y = (y % blockNorm_dims.y) +1;
    
    unsigned int i = threadIdx.x;
    unsigned int j = threadIdx.y;
    unsigned int k = threadIdx.z;
    unsigned int current_id = i + j*blockDim.x
                              + k*blockDim.x*blockDim.y;
    
    cache[current_id] = 0.;
    for (unsigned int i_=i ; i_ < blockHeightAndWidthInCells ; i_+=blockDim.x) {
        for (unsigned int j_=j ; j_ < blockHeightAndWidthInCells ; j_+=blockDim.y) {
            for (unsigned int k_=k ; k_ < numberOfOrientationBins ; k_+=blockDim.z) {
                unsigned int current_id_norm = i_ + j_*blockHeightAndWidthInCells
                              + k_*blockHeightAndWidthInCells*blockHeightAndWidthInCells
                              + (x-1)*blockHeightAndWidthInCells*blockHeightAndWidthInCells
                                *numberOfOrientationBins
                              + (y-1)*blockHeightAndWidthInCells*blockHeightAndWidthInCells
                                *numberOfOrientationBins*blockNorm_dims.x
                              + z*blockHeightAndWidthInCells*blockHeightAndWidthInCells
                                *numberOfOrientationBins*blockNorm_dims.x*blockNorm_dims.y;
                cache[current_id] += d_block[current_id_norm] * d_block[current_id_norm];
            }
        }
    }
    
    // Reduce operation
    // all threads in the current block have to compute d_blockNorm[x + hist2*y]
    int padding = blockDim.x*blockDim.y*blockDim.z/2;
    while (padding != 0) {
        if (current_id < padding)
            cache[current_id] += cache[current_id + padding];
        __syncthreads();
        padding /= 2;
    }
    
    // Several k values will have to participate to this value
    if (i == 0 && j == 0)
        d_blockNorm[x-1 + blockNorm_dims.x*(y-1)
                    + blockNorm_dims.x*blockNorm_dims.y*z] = cache[0];
}

__global__ void DalalTriggsHOGdescriptor_compute_outputImage(double *d_outputImage,
                                                             const double *d_block,
                                                             const double *d_blockNorm,
                                                             const dim3 blockNorm_dims,
                                                             const unsigned int numberOfOrientationBins,
                                                             const unsigned int blockHeightAndWidthInCells,
                                                             const unsigned int numWindows,
                                                             const unsigned int numberOfWindowsVertically,
                                                             const unsigned int block_size) {
    // Each thread has to compute one value of outputImage
    // for a given windows (blockIdx.x, blockIdx.y)
    
    // Compute window's index
    //unsigned int windowIndexHorizontal = blockIdx.x;
    //unsigned int windowIndexVertical = blockIdx.y;
    unsigned windowIndex = (blockIdx.y + numberOfWindowsVertically * blockIdx.x);
    
    // Retrieve ids of elements to compute during this thread
    // In most of the cases, the loop should be called only once
    for (unsigned int elementIndex(threadIdx.x) ; elementIndex < block_size ; elementIndex += blockDim.x) {
        //elementIndex = i + j*blockHeightAndWidthInCells
        //               + k*blockHeightAndWidthInCells*blockHeightAndWidthInCells
        //               + (x-1)*blockHeightAndWidthInCells*blockHeightAndWidthInCells
        //                 *numberOfOrientationBins
        //               + (y-1)*blockHeightAndWidthInCells*blockHeightAndWidthInCells
        //                 *numberOfOrientationBins*blockNorm_dims.x
        unsigned int i = elementIndex;
        unsigned int j = i / blockHeightAndWidthInCells;
        i %= blockHeightAndWidthInCells;
        unsigned int k = j / blockHeightAndWidthInCells;
        j %= blockHeightAndWidthInCells;
        unsigned int x = k / numberOfOrientationBins;
        k %= numberOfOrientationBins;
        unsigned int y = (x / blockNorm_dims.x) +1;
        x = (x % blockNorm_dims.x) +1;
        
        
        unsigned int descriptorIndex = k + numberOfOrientationBins * (
                j + blockHeightAndWidthInCells * (
                i + blockHeightAndWidthInCells * (
                y-1 + blockNorm_dims.y * (x-1))));
        
        double blockNorm = d_blockNorm[x-1 + blockNorm_dims.x*(y-1)
                           + blockNorm_dims.x*blockNorm_dims.y*windowIndex];
        if (blockNorm > 0) {
            blockNorm = sqrt(blockNorm);
            unsigned int current_id = elementIndex
                                      + windowIndex * blockHeightAndWidthInCells
                                        * blockHeightAndWidthInCells
                                        * numberOfOrientationBins
                                        * blockNorm_dims.x * blockNorm_dims.y;
            d_outputImage[windowIndex + numWindows*descriptorIndex]
                        = d_block[current_id] / blockNorm;
        } else
            d_outputImage[windowIndex + numWindows*descriptorIndex] = 0.;
    }
}
