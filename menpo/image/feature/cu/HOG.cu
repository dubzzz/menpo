#include "hip/hip_runtime.h"
#include "HOG.h"
#include "check_cuda_errors.hpp"

#define MAX_THREADS_1D 256
#define MAX_THREADS_2D  16

#define MAX_THREADS_3DX  4
#define MAX_THREADS_3DY  4
#define MAX_THREADS_3DZ 16

HOG::HOG(unsigned int windowHeight, unsigned int windowWidth,
         unsigned int numberOfChannels, unsigned int method,
         unsigned int numberOfOrientationBins,
         unsigned int cellHeightAndWidthInPixels,
         unsigned int blockHeightAndWidthInCells, bool enableSignedGradients,
         double l2normClipping) {
    unsigned int descriptorLengthPerBlock = 0,
                 numberOfBlocksPerWindowVertically = 0,
                 numberOfBlocksPerWindowHorizontally = 0;

    if (method == 1) {
        descriptorLengthPerBlock = blockHeightAndWidthInCells *
                                   blockHeightAndWidthInCells *
                                   numberOfOrientationBins;
        numberOfBlocksPerWindowVertically = 1 +
        (windowHeight - blockHeightAndWidthInCells*cellHeightAndWidthInPixels)
        / cellHeightAndWidthInPixels;
        numberOfBlocksPerWindowHorizontally = 1 +
        (windowWidth - blockHeightAndWidthInCells * cellHeightAndWidthInPixels)
        / cellHeightAndWidthInPixels;
    }
    else if (method==2) {
        descriptorLengthPerBlock = 27 + 4;
        numberOfBlocksPerWindowVertically =
        (unsigned int)round((double)windowHeight /
                            (double)cellHeightAndWidthInPixels) - 2;
        numberOfBlocksPerWindowHorizontally =
        (unsigned int)round((double)windowWidth /
                            (double)cellHeightAndWidthInPixels) - 2;
    }
    this->method = method;
    this->numberOfOrientationBins = numberOfOrientationBins;
    this->cellHeightAndWidthInPixels = cellHeightAndWidthInPixels;
    this->blockHeightAndWidthInCells = blockHeightAndWidthInCells;
    this->enableSignedGradients = enableSignedGradients;
    this->l2normClipping = l2normClipping;
    this->numberOfBlocksPerWindowHorizontally =
                    numberOfBlocksPerWindowHorizontally;
    this->numberOfBlocksPerWindowVertically =
                    numberOfBlocksPerWindowVertically;
    this->descriptorLengthPerBlock = descriptorLengthPerBlock;
    this->descriptorLengthPerWindow = numberOfBlocksPerWindowHorizontally *
                                      numberOfBlocksPerWindowVertically *
                                      descriptorLengthPerBlock;
    this->windowHeight = windowHeight;
    this->windowWidth = windowWidth;
    this->numberOfChannels = numberOfChannels;
}

HOG::~HOG() {
}


void HOG::apply(double *windowImage, double *descriptorVector) {
    if (this->method == 1)
        DalalTriggsHOGdescriptor(windowImage, this->numberOfOrientationBins,
                                 this->cellHeightAndWidthInPixels,
                                 this->blockHeightAndWidthInCells,
                                 this->enableSignedGradients,
                                 this->l2normClipping, this->windowHeight,
                                 this->windowWidth, this->numberOfChannels,
                                 descriptorVector);
    else
        ZhuRamananHOGdescriptor(windowImage, this->cellHeightAndWidthInPixels,
                                this->windowHeight, this->windowWidth,
                                this->numberOfChannels, descriptorVector);
}

// ZHU & RAMANAN: Face Detection, Pose Estimation and Landmark Localization
//                in the Wild
void ZhuRamananHOGdescriptor(double *inputImage,
                             int cellHeightAndWidthInPixels,
                             unsigned int imageHeight, unsigned int imageWidth,
                             unsigned int numberOfChannels,
                             double *descriptorMatrix) {
    // unit vectors used to compute gradient orientation
    double uu[9] = {1.0000, 0.9397, 0.7660, 0.500, 0.1736, -0.1736, -0.5000,
                    -0.7660, -0.9397};
    double vv[9] = {0.0000, 0.3420, 0.6428, 0.8660, 0.9848, 0.9848, 0.8660,
                    0.6428, 0.3420};
    int x, y, o;

    // memory for caching orientation histograms & their norms
    int blocks[2];
    blocks[0] = (int)round((double)imageHeight /
                           (double)cellHeightAndWidthInPixels);
    blocks[1] = (int)round((double)imageWidth /
                           (double)cellHeightAndWidthInPixels);
    double *hist = (double *)calloc(blocks[0] * blocks[1] * 18, sizeof(double));
    double *norm = (double *)calloc(blocks[0] * blocks[1], sizeof(double));

    // memory for HOG features
    int out[3];
    out[0] = max(blocks[0]-2, 0);
    out[1] = max(blocks[1]-2, 0);
    out[2] = 27+4;

    int visible[2];
    visible[0] = blocks[0] * cellHeightAndWidthInPixels;
    visible[1] = blocks[1] * cellHeightAndWidthInPixels;

    for (x = 1; x < visible[1] - 1; x++) {
        for (y = 1; y < visible[0] - 1; y++) {
            // compute gradient
            // first channel
            double *s = inputImage + min(x, imageWidth-2) * imageHeight +
                        min(y, imageHeight-2);
            double dy = *(s + 1) - *(s - 1);
            double dx = *(s + imageHeight) - *(s - imageHeight);
            double v = dx * dx + dy * dy;
            // rest of channels
            for (unsigned int z = 1; z < numberOfChannels; z++) {
                s += imageHeight * imageWidth;
                double dy2 = *(s + 1) - *(s - 1);
                double dx2 = *(s + imageHeight) - *(s - imageHeight);
                double v2 = dx2 * dx2 + dy2 * dy2;
                // pick channel with strongest gradient
                if (v2 > v) {
                    v = v2;
                    dx = dx2;
                    dy = dy2;
                }
            }

            // snap to one of 18 orientations
            double best_dot = 0;
            int best_o = 0;
            for (o = 0; o < 9; o++) {
                double dot = uu[o] * dx + vv[o] * dy;
                if (dot > best_dot) {
                    best_dot = dot;
                    best_o = o;
                }
                else if (-dot > best_dot) {
                    best_dot = - dot;
                    best_o = o + 9;
                }
            }

            // add to 4 histograms around pixel using linear interpolation
            double xp = ((double)x + 0.5) /
                        (double)cellHeightAndWidthInPixels - 0.5;
            double yp = ((double)y + 0.5) /
                        (double)cellHeightAndWidthInPixels - 0.5;
            int ixp = (int)floor(xp);
            int iyp = (int)floor(yp);
            double vx0 = xp - ixp;
            double vy0 = yp - iyp;
            double vx1 = 1.0 - vx0;
            double vy1 = 1.0 - vy0;
            v = sqrt(v);

            if (ixp >= 0 && iyp >= 0)
                *(hist + ixp*blocks[0] + iyp + best_o*blocks[0]*blocks[1])
                    += vx1 * vy1 * v;

            if (ixp+1 < blocks[1] && iyp >= 0)
                *(hist + (ixp+1)*blocks[0] + iyp + best_o*blocks[0]*blocks[1])
                    += vx0 * vy1 * v;

            if (ixp >= 0 && iyp+1 < blocks[0])
                *(hist + ixp*blocks[0] + (iyp+1) + best_o*blocks[0]*blocks[1])
                    += vx1 * vy0 * v;

            if (ixp+1 < blocks[1] && iyp+1 < blocks[0])
                *(hist + (ixp+1)*blocks[0] + (iyp+1) + best_o*blocks[0]*blocks[1])
                    += vx0 * vy0 * v;
        }
    }

    // compute energy in each block by summing over orientations
    for (int o = 0; o < 9; o++) {
        double *src1 = hist + o * blocks[0] * blocks[1];
        double *src2 = hist + (o + 9) * blocks[0] * blocks[1];
        double *dst = norm;
        double *end = norm + blocks[1] * blocks[0];
        while (dst < end) {
            *(dst++) += (*src1 + *src2) * (*src1 + *src2);
            src1++;
            src2++;
        }
    }

    // compute features
    for (x = 0; x < out[1]; x++) {
        for (y = 0; y < out[0]; y++) {
            double *dst = descriptorMatrix + x * out[0] + y;
            double *src, *p, n1, n2, n3, n4;

            p = norm + (x + 1) * blocks[0] + y + 1;
            n1 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks[0]) +
                            *(p + blocks[0] + 1) + eps);
            p = norm + (x + 1) * blocks[0] + y;
            n2 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks[0]) +
                            *(p + blocks[0] + 1) + eps);
            p = norm + x * blocks[0] + y + 1;
            n3 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks[0]) +
                            *(p + blocks[0] + 1) + eps);
            p = norm + x * blocks[0] + y;
            n4 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks[0]) +
                            *(p + blocks[0] + 1) + eps);

            double t1 = 0;
            double t2 = 0;
            double t3 = 0;
            double t4 = 0;

            // contrast-sensitive features
            src = hist + (x + 1) * blocks[0] + (y + 1);
            for (int o = 0; o < 18; o++) {
                double h1 = min(*src * n1, 0.2);
                double h2 = min(*src * n2, 0.2);
                double h3 = min(*src * n3, 0.2);
                double h4 = min(*src * n4, 0.2);
                *dst = 0.5 * (h1 + h2 + h3 + h4);
                t1 += h1;
                t2 += h2;
                t3 += h3;
                t4 += h4;
                dst += out[0] * out[1];
                src += blocks[0] * blocks[1];
            }

            // contrast-insensitive features
            src = hist + (x + 1) * blocks[0] + (y + 1);
            for (int o = 0; o < 9; o++) {
                double sum = *src + *(src + 9 * blocks[0] * blocks[1]);
                double h1 = min(sum * n1, 0.2);
                double h2 = min(sum * n2, 0.2);
                double h3 = min(sum * n3, 0.2);
                double h4 = min(sum * n4, 0.2);
                *dst = 0.5 * (h1 + h2 + h3 + h4);
                dst += out[0] * out[1];
                src += blocks[0] * blocks[1];
            }

            // texture features
            *dst = 0.2357 * t1;
            dst += out[0] * out[1];
            *dst = 0.2357 * t2;
            dst += out[0] * out[1];
            *dst = 0.2357 * t3;
            dst += out[0] * out[1];
            *dst = 0.2357 * t4;
        }
    }
    free(hist);
    free(norm);
}

__device__ double atomicAdd(double* address, double val) {
    // http://stackoverflow.com/questions/16882253/cuda-atomicadd-produces-wrong-result
    unsigned long long int* address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;
    do
    {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void DalalTriggsHOGdescriptor_precompute_histograms(double *d_h,
                                                               const dim3 h_dims,
                                                               const double *d_inputImage,
                                                               const unsigned int imageHeight,
                                                               const unsigned int imageWidth,
                                                               const unsigned int numberOfChannels,
                                                               const unsigned int numberOfOrientationBins,
                                                               const unsigned int cellHeightAndWidthInPixels,
                                                               const unsigned signedOrUnsignedGradients,
                                                               const double binsSize) {
    // Pre-compute histograms values
    // The array that contains "d_h" needs to be
    //  2*cellHeightAndWidthInPixels * 2*cellHeightAndWidthInPixels larger
    // Reduce kernel needs to be call in order to retrieve the expected
    // histogram
    
    // Retrieve pixel position
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int factor_a_dim = h_dims.x * h_dims.y * h_dims.z;
    unsigned int factor_z_dim = h_dims.x * h_dims.y;
    unsigned int factor_y_dim = h_dims.x;
     
    // Check if position is inside the image
    if (x >= imageWidth || y >= imageHeight)
        return;
    
    // Compute deltas
    double dx[3], dy[3];
    
    if (x == 0) {
        for (unsigned int z = 0; z < numberOfChannels; z++)
            dx[z] = d_inputImage[y + (x + 1) * imageHeight + z * imageHeight * imageWidth];
    } else {
        if (x == imageWidth - 1) {
            for (unsigned int z = 0; z < numberOfChannels; z++)
                dx[z] = -d_inputImage[y + (x - 1) * imageHeight + z * imageHeight * imageWidth];
        } else {
            for (unsigned int z = 0; z < numberOfChannels; z++)
                dx[z] = d_inputImage[y + (x + 1) * imageHeight + z * imageHeight * imageWidth] - d_inputImage[y + (x - 1) * imageHeight + z * imageHeight * imageWidth];
        }
    }

    if(y == 0) {
        for (unsigned int z = 0; z < numberOfChannels; z++)
            dy[z] = -d_inputImage[y + 1 + x * imageHeight + z * imageHeight * imageWidth];
    } else {
        if (y == imageHeight - 1) {
            for (unsigned int z = 0; z < numberOfChannels; z++)
                dy[z] = d_inputImage[y - 1 + x * imageHeight + z * imageHeight * imageWidth];
        } else {
            for (unsigned int z = 0; z < numberOfChannels; z++)
                dy[z] = -d_inputImage[y + 1 + x * imageHeight + z * imageHeight * imageWidth] + d_inputImage[y - 1 + x * imageHeight + z * imageHeight * imageWidth];
        }
    }

    // Choose dominant channel based on magnitude
    double gradientMagnitude = sqrt(dx[0] * dx[0] + dy[0] * dy[0]);
    double gradientOrientation = atan2(dy[0], dx[0]);
    if (numberOfChannels > 1) {
        double tempMagnitude = gradientMagnitude;
        for (unsigned int cli = 1 ; cli < numberOfChannels ; ++cli) {
            tempMagnitude= sqrt(dx[cli] * dx[cli] + dy[cli] * dy[cli]);
            if (tempMagnitude > gradientMagnitude) {
                gradientMagnitude = tempMagnitude;
                gradientOrientation = atan2(dy[cli], dx[cli]);
            }
        }
    }

    if (gradientOrientation < 0)
        gradientOrientation += pi + (signedOrUnsignedGradients == 1) * pi;

    // Trilinear interpolation
    int bin1 = (gradientOrientation / binsSize) - 1;
    unsigned int bin2 = bin1 + 1;
    int x1   = x / cellHeightAndWidthInPixels;
    int x2   = x1 + 1;
    int y1   = y / cellHeightAndWidthInPixels;
    int y2   = y1 + 1;
    int a    = (x % (2*cellHeightAndWidthInPixels)) * (2*cellHeightAndWidthInPixels)
               + (y % (2*cellHeightAndWidthInPixels));
    
    double Xc = (x1 + 1 - 1.5) * cellHeightAndWidthInPixels + 0.5;
    double Yc = (y1 + 1 - 1.5) * cellHeightAndWidthInPixels + 0.5;
    double Oc = (bin1 + 1 + 1 - 1.5) * binsSize;
    
    if (bin2 == numberOfOrientationBins)
        bin2 = 0;
    
    if (bin1 < 0)
        bin1 = numberOfOrientationBins - 1;
    
    // Compute histograms
    //  using reduce-pattern
    
    // d_h contains (2*cellHeightAndWidthInPixels)*(2*cellHeightAndWidthInPixels)
    //  times required d_h
    //  which is equal to: 2*8*2*8 = 256
    // d_h needs to be set to 0.
    
    d_h[y1 + x1*factor_y_dim + bin1*factor_z_dim + a*factor_a_dim] = gradientMagnitude *
                                                        (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                                                        (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                                                        (1-((gradientOrientation-Oc)/binsSize));
    d_h[y1 + x1*factor_y_dim + bin2*factor_z_dim + a*factor_a_dim] = gradientMagnitude *
                                                        (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                                                        (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                                                        (((gradientOrientation-Oc)/binsSize));
    d_h[y2 + x1*factor_y_dim + bin1*factor_z_dim + a*factor_a_dim] = gradientMagnitude *
                                                        (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                                                        (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                                                        (1-((gradientOrientation-Oc)/binsSize));
    d_h[y2 + x1*factor_y_dim + bin2*factor_z_dim + a*factor_a_dim] = gradientMagnitude *
                                                        (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                                                        (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                                                        (((gradientOrientation-Oc)/binsSize));
    d_h[y1 + x2*factor_y_dim + bin1*factor_z_dim + a*factor_a_dim] = gradientMagnitude *
                                                        (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                                                        (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                                                        (1-((gradientOrientation-Oc)/binsSize));
    d_h[y1 + x2*factor_y_dim + bin2*factor_z_dim + a*factor_a_dim] = gradientMagnitude *
                                                        (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                                                        (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                                                        (((gradientOrientation-Oc)/binsSize));
    d_h[y2 + x2*factor_y_dim + bin1*factor_z_dim + a*factor_a_dim] = gradientMagnitude *
                                                        (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                                                        (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                                                        (1-((gradientOrientation-Oc)/binsSize));
    d_h[y2 + x2*factor_y_dim + bin2*factor_z_dim + a*factor_a_dim] = gradientMagnitude *
                                                        (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                                                        (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                                                        (((gradientOrientation-Oc)/binsSize));
}

__global__ void DalalTriggsHOGdescriptor_reduce_histograms(double *d_h,
                                                           const dim3 h_dims,
                                                           const unsigned int cellHeightAndWidthInPixels) {
    // cache size has to be a power of 2
    // usually set to MAX_THREADS_1D
    extern __shared__ double cache[];
    
    // Compute factors
    unsigned int factor_a_dim = h_dims.x * h_dims.y * h_dims.z;
    unsigned int factor_z_dim = h_dims.x * h_dims.y;
    unsigned int factor_y_dim = h_dims.x;
    
    // Retrieve indice of the element
    unsigned int x = blockIdx.y;
    unsigned int y = blockIdx.x;
    unsigned int bin = blockIdx.z;
    unsigned int a = threadIdx.x;
    unsigned int h_element_id = y + x*factor_y_dim + bin*factor_z_dim;
    unsigned int numElements = (2*cellHeightAndWidthInPixels) * (2*cellHeightAndWidthInPixels);
    
    // Copy to cache
    // The forloop make it possible to deal with too many data:
    //  in that case, a thread could have to deal with more than one element
    // /!\ Not Coalesced Memory access - can be slow
    if (a < numElements)
    {
        cache[a] = d_h[h_element_id + a*factor_a_dim];
        for (unsigned int a_(a+blockDim.x) ; a_ < numElements ; a_ += blockDim.x)
            cache[a] += d_h[h_element_id + a_*factor_a_dim];
    }
    else
        cache[a] = 0.;
    __syncthreads();
    
    // Reduce operation
    // all threads in the current block have to compute d_h[h_element_id]
    int padding = blockDim.x/2;
    while (padding != 0) {
        if (a < padding)
            cache[a] += cache[a + padding];
        __syncthreads();
        padding /= 2;
    }
    
    // Copy to d_h[h_element_id]
    if (a == 0)
        d_h[h_element_id] = cache[0];
}

__global__ void DalalTriggsHOGdescriptor_compute_blocknorm(double *d_blockNorm,
                                                           const dim3 blockNorm_dims,
                                                           const double *d_h,
                                                           const dim3 h_dims,
                                                           const unsigned int numberOfOrientationBins,
                                                           const unsigned int blockHeightAndWidthInCells) {
    // 2D-reduce to compute d_blockNorm for every (x,y)
    
    // Size of shared memory must be blockDim.x*blockDim.y*blockDim.z
    // and a power of 2
    extern __shared__ double cache[];
    
    // Compute factors
    unsigned int factor_z_dim = h_dims.x * h_dims.y;
    unsigned int factor_y_dim = h_dims.x;
    
    // Retrieve indice of the element
    unsigned int x = blockIdx.x +1;
    unsigned int y = blockIdx.y +1;
    
    unsigned int i = threadIdx.x;
    unsigned int j = threadIdx.y;
    unsigned int k = threadIdx.z;
    unsigned int current_id = i + j*blockDim.x
                              + k*blockDim.x*blockDim.y;
    
    cache[current_id] = 0.;
    for (unsigned int i_=i ; i_ < blockHeightAndWidthInCells ; i_+=blockDim.x)
        for (unsigned int j_=j ; j_ < blockHeightAndWidthInCells ; j_+=blockDim.y)
            for (unsigned int k_=k ; k_ < numberOfOrientationBins ; k_+=blockDim.z)
                cache[current_id] += d_h[y+i_ + (x+j_) * factor_y_dim + k_ * factor_z_dim]
                                     * d_h[y+i_ + (x+j_) * factor_y_dim + k_ * factor_z_dim];
    
    // Reduce operation
    // all threads in the current block have to compute d_blockNorm[x + hist2*y]
    int padding = blockDim.x*blockDim.y*blockDim.z/2;
    while (padding != 0) {
        if (current_id < padding)
            cache[current_id] += cache[current_id + padding];
        __syncthreads();
        padding /= 2;
    }
    
    // Several k values will have to participate to this value
    if (i == 0 && j == 0)
        d_blockNorm[x-1 + blockNorm_dims.x*(y-1)] = cache[0];
}

__global__ void DalalTriggsHOGdescriptor_compute_block(double *d_block,
                                                       const double *d_blockNorm,
                                                       const dim3 blockNorm_dims,
                                                       const double *d_h,
                                                       const dim3 h_dims,
                                                       const unsigned int numberOfOrientationBins,
                                                       const unsigned int blockHeightAndWidthInCells,
                                                       const double l2normClipping) {
    // Each thread has to compute one value of block[i,j,k,x,y]
    
    // Compute factors
    unsigned int factor_z_dim = h_dims.x * h_dims.y;
    unsigned int factor_y_dim = h_dims.x;
    
    // Retrieve indice of the element
    unsigned int x = blockIdx.x;
    unsigned int y = blockIdx.y;
    
    unsigned int i = threadIdx.x;
    if (x >= blockNorm_dims.x) {
        i += ((unsigned int) (x/blockNorm_dims.x)) * blockDim.x;
        x = x % blockNorm_dims.x;
    }
    x += 1;
    unsigned int j = threadIdx.y;
    if (y >= blockNorm_dims.y) {
        j += ((unsigned int) (y/blockNorm_dims.y)) * blockDim.y;
        y = y % blockNorm_dims.y;
    }
    y += 1;
    unsigned int k = threadIdx.z + blockIdx.z * blockDim.z;
    
    if (i >= blockHeightAndWidthInCells || j >= blockHeightAndWidthInCells
        || k >= numberOfOrientationBins)
        return;
    
    unsigned int current_id = i + j*blockHeightAndWidthInCells
                              + k*blockHeightAndWidthInCells*blockHeightAndWidthInCells
                              + (x-1)*blockHeightAndWidthInCells*blockHeightAndWidthInCells
                                *numberOfOrientationBins
                              + (y-1)*blockHeightAndWidthInCells*blockHeightAndWidthInCells
                                *numberOfOrientationBins*blockNorm_dims.x;
    
    
    double blockNorm = sqrt(d_blockNorm[x-1 + blockNorm_dims.x*(y-1)]);
    if (blockNorm > 0) {
        double tmpValue = d_h[y+i + (x+j) * factor_y_dim + k * factor_z_dim] / blockNorm;
        if (tmpValue > l2normClipping)
            d_block[current_id] = l2normClipping;
        else
            d_block[current_id] = tmpValue;
    } else
        d_block[current_id] = 0.;
}

__global__ void DalalTriggsHOGdescriptor_compute_blocknorm2(double *d_blockNorm,
                                                            const dim3 blockNorm_dims,
                                                            const double *d_block,
                                                            const unsigned int numberOfOrientationBins,
                                                            const unsigned int blockHeightAndWidthInCells) {
    // 2D-reduce to compute d_blockNorm for every (x,y)
    
    // Size of shared memory must be blockDim.x*blockDim.y*blockDim.z
    // and a power of 2
    extern __shared__ double cache[];
    
    // Retrieve indice of the element
    unsigned int x = blockIdx.x +1;
    unsigned int y = blockIdx.y +1;
    
    unsigned int i = threadIdx.x;
    unsigned int j = threadIdx.y;
    unsigned int k = threadIdx.z;
    unsigned int current_id = i + j*blockDim.x
                              + k*blockDim.x*blockDim.y;
    
    cache[current_id] = 0.;
    for (unsigned int i_=i ; i_ < blockHeightAndWidthInCells ; i_+=blockDim.x) {
        for (unsigned int j_=j ; j_ < blockHeightAndWidthInCells ; j_+=blockDim.y) {
            for (unsigned int k_=k ; k_ < numberOfOrientationBins ; k_+=blockDim.z) {
                unsigned int current_id_norm = i_ + j_*blockHeightAndWidthInCells
                              + k_*blockHeightAndWidthInCells*blockHeightAndWidthInCells
                              + (x-1)*blockHeightAndWidthInCells*blockHeightAndWidthInCells
                                *numberOfOrientationBins
                              + (y-1)*blockHeightAndWidthInCells*blockHeightAndWidthInCells
                                *numberOfOrientationBins*blockNorm_dims.x;
                cache[current_id] += d_block[current_id_norm] * d_block[current_id_norm];
            }
        }
    }
    
    // Reduce operation
    // all threads in the current block have to compute d_blockNorm[x + hist2*y]
    int padding = blockDim.x*blockDim.y*blockDim.z/2;
    while (padding != 0) {
        if (current_id < padding)
            cache[current_id] += cache[current_id + padding];
        __syncthreads();
        padding /= 2;
    }
    
    // Several k values will have to participate to this value
    if (i == 0 && j == 0)
        d_blockNorm[x-1 + blockNorm_dims.x*(y-1)] = cache[0];
}

// DALAL & TRIGGS: Histograms of Oriented Gradients for Human Detection
void DalalTriggsHOGdescriptor(double *inputImage,
                              unsigned int numberOfOrientationBins,
                              unsigned int cellHeightAndWidthInPixels,
                              unsigned int blockHeightAndWidthInCells,
                              bool signedOrUnsignedGradientsBool,
                              double l2normClipping, unsigned int imageHeight,
                              unsigned int imageWidth,
                              unsigned int numberOfChannels,
                              double *descriptorVector) {
   
    // ** VARIABLES **
    //  * Compute gradients & Compute histograms
    
    const int hist1 = 2 + (imageHeight / cellHeightAndWidthInPixels);
    const int hist2 = 2 + (imageWidth / cellHeightAndWidthInPixels);
    const dim3 h_dims(hist1, hist2, numberOfOrientationBins);
    double *d_h = NULL, *d_inputImage = NULL;
    const dim3 dimBlock(MAX_THREADS_2D, MAX_THREADS_2D, 1);
    const dim3 dimGrid((imageWidth + dimBlock.x -1)/dimBlock.x, (imageHeight + dimBlock.y -1)/dimBlock.y, 1);
    const unsigned int numCopies_d_h = (2*cellHeightAndWidthInPixels) * (2*cellHeightAndWidthInPixels);
     
    // each block is responsible to evaluate the value of
    // d_h[blockIdx.x][blockIdx.y][blockIdx.z]
    // the reduce operation concerns (2*cellHeightAndWidthInPixels) * (2*cellHeightAndWidthInPixels)
    // elements
    const dim3 dimGrid_reduce(h_dims.x, h_dims.y, h_dims.z);
    
    //  * Block normalization
    
    int descriptorIndex(0);
    
    // Each block has to compute its blockNorm[x][y]
    // stored into d_blockNorm[x-1 + blockNorm_dims.x*(y-1)]
    // Computation done with reduce-pattern
    //   x: unsigned int x = 1; x < hist2 - blockHeightAndWidthInCells; x++
    //      => requires (hist2 - blockHeightAndWidthInCells -1) Blocks
    //   y: unsigned int y = 1; y < hist1 - blockHeightAndWidthInCells; y++
    //      => requires (hist1 - blockHeightAndWidthInCells -1) Blocks
    // Each block works on 3D-Threads
    //   i: unsigned int i = 0; i < blockHeightAndWidthInCells; i++
    //   j: unsigned int j = 0; j < blockHeightAndWidthInCells; j++
    //      => usually: blockHeightAndWidthInCells=2
    //   k: unsigned int k = 0; k < numberOfOrientationBins; k++
    //      => usually: larger than blockHeightAndWidthInCells
    double *d_blockNorm = NULL;
    const dim3 blockNorm_dims(hist2 - blockHeightAndWidthInCells -1,
                              hist1 - blockHeightAndWidthInCells -1);
    const dim3 dimBlock_norm(MAX_THREADS_3DX, MAX_THREADS_3DY, MAX_THREADS_3DZ);
    const dim3 dimGrid_norm(blockNorm_dims.x, blockNorm_dims.y, 1);
    double h_blockNorm[blockNorm_dims.x * blockNorm_dims.y];
    
    // Each thread has to compute one value of block[i,j,k,x,y]
    // The corresponding is stored into
    //   unsigned int current_id = i + j*blockHeightAndWidthInCells
    //      + k*blockHeightAndWidthInCells*blockHeightAndWidthInCells
    //      + (x-1)*blockHeightAndWidthInCells*blockHeightAndWidthInCells
    //        *numberOfOrientationBins
    //      + (y-1)*blockHeightAndWidthInCells*blockHeightAndWidthInCells
    //        *numberOfOrientationBins*blockNorm_dims.x;
    //   d_block[current_id]
    // The number of blocks in the grid depends on blockNorm_dims,
    // numberOfOrientationBins and blockHeightAndWidthInCells
    double *d_block = NULL;
    double h_block[cellHeightAndWidthInPixels * cellHeightAndWidthInPixels
                   * numberOfOrientationBins * blockNorm_dims.x
                   * blockNorm_dims.y];
    const dim3 dimBlock_block(MAX_THREADS_3DX, MAX_THREADS_3DY, MAX_THREADS_3DZ);
    const dim3 dimGrid_block(
            blockNorm_dims.x
            * ((blockHeightAndWidthInCells+MAX_THREADS_3DX-1)/MAX_THREADS_3DX),
            blockNorm_dims.y
            * ((blockHeightAndWidthInCells+MAX_THREADS_3DY-1)/MAX_THREADS_3DY),
            (numberOfOrientationBins + MAX_THREADS_3DZ -1)/MAX_THREADS_3DZ);
    
    // ** GRADIENTS and HISTOGRAMS **
    // Compute gradients (zero padding)
    // Compute histograms
    //  using CUDA
    
    // d_h is set numCopies_d_h times larger in order to benefit from reduce pattern
    cudaErrorCheck_goto(hipMalloc(&d_h, h_dims.x * h_dims.y * h_dims.z * numCopies_d_h * sizeof(double)));
    cudaErrorCheck_goto(hipMemset(d_h, 0., h_dims.x * h_dims.y * h_dims.z * numCopies_d_h * sizeof(double)));
    
    cudaErrorCheck_goto(hipMalloc(&d_inputImage, imageHeight * imageWidth * numberOfChannels * sizeof(double)));
    cudaErrorCheck_goto(hipMemcpy(d_inputImage, inputImage, imageHeight * imageWidth * numberOfChannels * sizeof(double), hipMemcpyHostToDevice));
    
    DalalTriggsHOGdescriptor_precompute_histograms<<<dimGrid, dimBlock>>>(d_h, h_dims,
                                                                          d_inputImage, imageHeight, imageWidth, numberOfChannels,
                                                                          numberOfOrientationBins, cellHeightAndWidthInPixels,
                                                                          signedOrUnsignedGradientsBool ? 1 : 0 /*signedOrUnsignedGradients*/,
                                                                          (1 + (signedOrUnsignedGradientsBool ? 1 : 0)) * pi / numberOfOrientationBins /*binsSize*/);
    cudaErrorCheck_goto(hipDeviceSynchronize()); // block until the device is finished
    
    DalalTriggsHOGdescriptor_reduce_histograms<<<dimGrid_reduce,
                                                 MAX_THREADS_1D,
                                                 MAX_THREADS_1D*sizeof(double)>>>
                                                    (d_h, h_dims,
                                                     cellHeightAndWidthInPixels);
    cudaErrorCheck_goto(hipDeviceSynchronize()); // block until the device is finished
    
    cudaErrorCheck_goto(hipFree(d_inputImage));
    d_inputImage = NULL;
    
    // ** BLOCK NORMALIZATION **
    
    // Evaluate blockNorm based on d_h
    cudaErrorCheck_goto(hipMalloc(&d_blockNorm, blockNorm_dims.x * blockNorm_dims.y * sizeof(double)));
    cudaErrorCheck_goto(hipMemset(d_blockNorm, 0., blockNorm_dims.x * blockNorm_dims.y * sizeof(double)));
    
    DalalTriggsHOGdescriptor_compute_blocknorm<<<dimGrid_norm,
                                                 dimBlock_norm,
                                                 MAX_THREADS_3DX
                                                 * MAX_THREADS_3DY
                                                 * MAX_THREADS_3DZ
                                                 * sizeof(double)>>>
                                                    (d_blockNorm, blockNorm_dims,
                                                     d_h, h_dims,
                                                     numberOfOrientationBins,
                                                     cellHeightAndWidthInPixels);
    cudaErrorCheck_goto(hipDeviceSynchronize()); // block until the device is finished
    
    // Compute block[i,j,k,x,y]
    cudaErrorCheck_goto(hipMalloc(&d_block, cellHeightAndWidthInPixels
                                             * cellHeightAndWidthInPixels
                                             * numberOfOrientationBins
                                             * blockNorm_dims.x
                                             * blockNorm_dims.y
                                             * sizeof(double)));
    
    DalalTriggsHOGdescriptor_compute_block<<<dimGrid_block,
                                             dimBlock_block>>>
                                                (d_block,
                                                 d_blockNorm, blockNorm_dims,
                                                 d_h, h_dims,
                                                 numberOfOrientationBins,
                                                 blockHeightAndWidthInCells,
                                                 l2normClipping);
    cudaErrorCheck_goto(hipDeviceSynchronize());
    
    cudaErrorCheck_goto(hipFree(d_h));
    d_h = NULL;
    
    // Evaluate blockNorm based on d_block
    DalalTriggsHOGdescriptor_compute_blocknorm2<<<dimGrid_norm,
                                                  dimBlock_norm,
                                                  MAX_THREADS_3DX
                                                  * MAX_THREADS_3DY
                                                  * MAX_THREADS_3DZ
                                                  * sizeof(double)>>>
                                                    (d_blockNorm, blockNorm_dims,
                                                     d_block,
                                                     numberOfOrientationBins,
                                                     blockHeightAndWidthInCells);
    cudaErrorCheck_goto(hipDeviceSynchronize()); // block until the device is finished
    
    // Copy to CPU
    cudaErrorCheck_goto(hipMemcpy(h_blockNorm, d_blockNorm,
                                   blockNorm_dims.x * blockNorm_dims.y * sizeof(double),
                                   hipMemcpyDeviceToHost));
    
    cudaErrorCheck_goto(hipMemcpy(h_block, d_block,
                                   cellHeightAndWidthInPixels
                                   * cellHeightAndWidthInPixels
                                   * numberOfOrientationBins
                                   * blockNorm_dims.x * blockNorm_dims.y
                                   * sizeof(double),
                                   hipMemcpyDeviceToHost));
    
    cudaErrorCheck_goto(hipFree(d_blockNorm));
    d_blockNorm = NULL;
    cudaErrorCheck_goto(hipFree(d_block));
    d_block = NULL;
    
    for (unsigned int x = 1; x < hist2 - blockHeightAndWidthInCells; x++) {
        for (unsigned int y = 1; y < hist1 - blockHeightAndWidthInCells; y++) {
            float blockNorm(sqrt(h_blockNorm[x-1 + blockNorm_dims.x*(y-1)]));
            for (unsigned int i = 0; i < blockHeightAndWidthInCells; i++) {
                for (unsigned int j = 0; j < blockHeightAndWidthInCells; j++) {
                    for (unsigned int k = 0; k < numberOfOrientationBins; k++) {
                        unsigned int current_id = i + j*blockHeightAndWidthInCells
                            + k*blockHeightAndWidthInCells*blockHeightAndWidthInCells
                            + (x-1)*blockHeightAndWidthInCells*blockHeightAndWidthInCells
                                *numberOfOrientationBins
                            + (y-1)*blockHeightAndWidthInCells*blockHeightAndWidthInCells
                                *numberOfOrientationBins*blockNorm_dims.x;
                        if (blockNorm > 0)
                            descriptorVector[descriptorIndex] =
                                h_block[current_id] / blockNorm;
                        else
                            descriptorVector[descriptorIndex] = 0.0;
                        descriptorIndex++;
                    }
                }
            }
        }
    }
    return;

onfailure:
    if (d_h != NULL)
        hipFree(d_h);
    if (d_blockNorm != NULL)
        hipFree(d_blockNorm);
    if (d_block != NULL)
        hipFree(d_block);
    if (d_inputImage != NULL)
        hipFree(d_inputImage);
    
    return;
}
