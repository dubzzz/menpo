#include "hip/hip_runtime.h"
#include "HOG.h"
#include "check_cuda_errors.hpp"
#include "Python.h"

#define MAX_THREADS_1D 256
#define MAX_THREADS_2D  16

/* Kernels' signature declaration */

__global__ void DalalTriggsHOGdescriptor_compute_histograms(double *d_h,
                                                            const dim3 h_dims,
                                                            const double *d_inputImage,
                                                            const unsigned int imageHeight,
                                                            const unsigned int imageWidth,
                                                            const unsigned int windowHeight,
                                                            const unsigned int windowWidth,
                                                            const unsigned int numberOfChannels,
                                                            const unsigned int numberOfOrientationBins,
                                                            const unsigned int cellHeightAndWidthInPixels,
                                                            const unsigned signedOrUnsignedGradients,
                                                            const double binsSize,
                                                            const int numHistograms,
                                                            const int numberOfWindowsVertically,
                                                            const int numberOfWindowsHorizontally,
                                                            const bool enablePadding,
                                                            const int windowStepVertical, const int windowStepHorizontal);

/* HOG methods */

HOG::HOG(unsigned int windowHeight, unsigned int windowWidth,
         unsigned int numberOfChannels, unsigned int method,
         unsigned int numberOfOrientationBins,
         unsigned int cellHeightAndWidthInPixels,
         unsigned int blockHeightAndWidthInCells, bool enableSignedGradients,
         double l2normClipping) {
    unsigned int descriptorLengthPerBlock = 0,
                 numberOfBlocksPerWindowVertically = 0,
                 numberOfBlocksPerWindowHorizontally = 0;

    if (method == 1) {
        descriptorLengthPerBlock = blockHeightAndWidthInCells *
                                   blockHeightAndWidthInCells *
                                   numberOfOrientationBins;
        numberOfBlocksPerWindowVertically = 1 +
        (windowHeight - blockHeightAndWidthInCells*cellHeightAndWidthInPixels)
        / cellHeightAndWidthInPixels;
        numberOfBlocksPerWindowHorizontally = 1 +
        (windowWidth - blockHeightAndWidthInCells * cellHeightAndWidthInPixels)
        / cellHeightAndWidthInPixels;
    }
    else if (method==2) {
        descriptorLengthPerBlock = 27 + 4;
        numberOfBlocksPerWindowVertically =
        (unsigned int)round((double)windowHeight /
                            (double)cellHeightAndWidthInPixels) - 2;
        numberOfBlocksPerWindowHorizontally =
        (unsigned int)round((double)windowWidth /
                            (double)cellHeightAndWidthInPixels) - 2;
    }
    this->method = method;
    this->numberOfOrientationBins = numberOfOrientationBins;
    this->cellHeightAndWidthInPixels = cellHeightAndWidthInPixels;
    this->blockHeightAndWidthInCells = blockHeightAndWidthInCells;
    this->enableSignedGradients = enableSignedGradients;
    this->l2normClipping = l2normClipping;
    this->numberOfBlocksPerWindowHorizontally =
                    numberOfBlocksPerWindowHorizontally;
    this->numberOfBlocksPerWindowVertically =
                    numberOfBlocksPerWindowVertically;
    this->descriptorLengthPerBlock = descriptorLengthPerBlock;
    this->descriptorLengthPerWindow = numberOfBlocksPerWindowHorizontally *
                                      numberOfBlocksPerWindowVertically *
                                      descriptorLengthPerBlock;
    this->windowHeight = windowHeight;
    this->windowWidth = windowWidth;
    this->numberOfChannels = numberOfChannels;
}

HOG::~HOG() {
}


void HOG::applyOnChunk(double *windowImage, double *descriptorVector) {
    if (this->method == 1)
        PyErr_SetString(PyExc_RuntimeError,
                        "HOG::applyOnChunk is not implemented for DalalTriggs");
    else
        ZhuRamananHOGdescriptor(windowImage, this->cellHeightAndWidthInPixels,
                                this->windowHeight, this->windowWidth,
                                this->numberOfChannels, descriptorVector);
}

void HOG::applyOnImage(const ImageWindowIterator &iwi, const double *image,
                       double *outputImage, int *windowsCenters) {
    __CLOG__
    double *d_image = 0;
    if (this->method == 1) {
        const unsigned int imageHeight = iwi._imageHeight;
        const unsigned int imageWidth = iwi._imageWidth;
        const unsigned int numberOfChannels = iwi._numberOfChannels;
        
        __START__
        cudaErrorCheck_goto(hipMalloc(&d_image, imageHeight * imageWidth * numberOfChannels * sizeof(double)));
        cudaErrorCheck_goto(hipMemcpy(d_image, image, imageHeight * imageWidth * numberOfChannels * sizeof(double), hipMemcpyHostToDevice));
        __STOP("@ Malloc & Memcpy for <image> @")
        this->DalalTriggsHOGdescriptorOnImage(iwi, d_image, outputImage, windowsCenters);
        __START__
        cudaErrorCheck_goto(hipFree(d_image));
        d_image = 0;
        __STOP("@ Free for <image> @")
    } else
        PyErr_SetString(PyExc_RuntimeError,
                        "HOG::applyOnImage is not implemented for ZhuRamanan");
    return;

onfailure:
    hipFree(d_image);
    return;
}

bool HOG::isApplyOnImage() {
    if (this->method == 1) // easier to read this way
        return true;
    else
        return false;
}

// ZHU & RAMANAN: Face Detection, Pose Estimation and Landmark Localization
//                in the Wild
void ZhuRamananHOGdescriptor(double *inputImage,
                             int cellHeightAndWidthInPixels,
                             unsigned int imageHeight, unsigned int imageWidth,
                             unsigned int numberOfChannels,
                             double *descriptorMatrix) {
    // unit vectors used to compute gradient orientation
    double uu[9] = {1.0000, 0.9397, 0.7660, 0.500, 0.1736, -0.1736, -0.5000,
                    -0.7660, -0.9397};
    double vv[9] = {0.0000, 0.3420, 0.6428, 0.8660, 0.9848, 0.9848, 0.8660,
                    0.6428, 0.3420};
    int x, y, o;

    // memory for caching orientation histograms & their norms
    int blocks[2];
    blocks[0] = (int)round((double)imageHeight /
                           (double)cellHeightAndWidthInPixels);
    blocks[1] = (int)round((double)imageWidth /
                           (double)cellHeightAndWidthInPixels);
    double *hist = (double *)calloc(blocks[0] * blocks[1] * 18, sizeof(double));
    double *norm = (double *)calloc(blocks[0] * blocks[1], sizeof(double));

    // memory for HOG features
    int out[3];
    out[0] = max(blocks[0]-2, 0);
    out[1] = max(blocks[1]-2, 0);
    out[2] = 27+4;

    int visible[2];
    visible[0] = blocks[0] * cellHeightAndWidthInPixels;
    visible[1] = blocks[1] * cellHeightAndWidthInPixels;

    for (x = 1; x < visible[1] - 1; x++) {
        for (y = 1; y < visible[0] - 1; y++) {
            // compute gradient
            // first channel
            double *s = inputImage + min(x, imageWidth-2) * imageHeight +
                        min(y, imageHeight-2);
            double dy = *(s + 1) - *(s - 1);
            double dx = *(s + imageHeight) - *(s - imageHeight);
            double v = dx * dx + dy * dy;
            // rest of channels
            for (unsigned int z = 1; z < numberOfChannels; z++) {
                s += imageHeight * imageWidth;
                double dy2 = *(s + 1) - *(s - 1);
                double dx2 = *(s + imageHeight) - *(s - imageHeight);
                double v2 = dx2 * dx2 + dy2 * dy2;
                // pick channel with strongest gradient
                if (v2 > v) {
                    v = v2;
                    dx = dx2;
                    dy = dy2;
                }
            }

            // snap to one of 18 orientations
            double best_dot = 0;
            int best_o = 0;
            for (o = 0; o < 9; o++) {
                double dot = uu[o] * dx + vv[o] * dy;
                if (dot > best_dot) {
                    best_dot = dot;
                    best_o = o;
                }
                else if (-dot > best_dot) {
                    best_dot = - dot;
                    best_o = o + 9;
                }
            }

            // add to 4 histograms around pixel using linear interpolation
            double xp = ((double)x + 0.5) /
                        (double)cellHeightAndWidthInPixels - 0.5;
            double yp = ((double)y + 0.5) /
                        (double)cellHeightAndWidthInPixels - 0.5;
            int ixp = (int)floor(xp);
            int iyp = (int)floor(yp);
            double vx0 = xp - ixp;
            double vy0 = yp - iyp;
            double vx1 = 1.0 - vx0;
            double vy1 = 1.0 - vy0;
            v = sqrt(v);

            if (ixp >= 0 && iyp >= 0)
                *(hist + ixp*blocks[0] + iyp + best_o*blocks[0]*blocks[1])
                    += vx1 * vy1 * v;

            if (ixp+1 < blocks[1] && iyp >= 0)
                *(hist + (ixp+1)*blocks[0] + iyp + best_o*blocks[0]*blocks[1])
                    += vx0 * vy1 * v;

            if (ixp >= 0 && iyp+1 < blocks[0])
                *(hist + ixp*blocks[0] + (iyp+1) + best_o*blocks[0]*blocks[1])
                    += vx1 * vy0 * v;

            if (ixp+1 < blocks[1] && iyp+1 < blocks[0])
                *(hist + (ixp+1)*blocks[0] + (iyp+1) + best_o*blocks[0]*blocks[1])
                    += vx0 * vy0 * v;
        }
    }

    // compute energy in each block by summing over orientations
    for (int o = 0; o < 9; o++) {
        double *src1 = hist + o * blocks[0] * blocks[1];
        double *src2 = hist + (o + 9) * blocks[0] * blocks[1];
        double *dst = norm;
        double *end = norm + blocks[1] * blocks[0];
        while (dst < end) {
            *(dst++) += (*src1 + *src2) * (*src1 + *src2);
            src1++;
            src2++;
        }
    }

    // compute features
    for (x = 0; x < out[1]; x++) {
        for (y = 0; y < out[0]; y++) {
            double *dst = descriptorMatrix + x * out[0] + y;
            double *src, *p, n1, n2, n3, n4;

            p = norm + (x + 1) * blocks[0] + y + 1;
            n1 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks[0]) +
                            *(p + blocks[0] + 1) + eps);
            p = norm + (x + 1) * blocks[0] + y;
            n2 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks[0]) +
                            *(p + blocks[0] + 1) + eps);
            p = norm + x * blocks[0] + y + 1;
            n3 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks[0]) +
                            *(p + blocks[0] + 1) + eps);
            p = norm + x * blocks[0] + y;
            n4 = 1.0 / sqrt(*p + *(p + 1) + *(p + blocks[0]) +
                            *(p + blocks[0] + 1) + eps);

            double t1 = 0;
            double t2 = 0;
            double t3 = 0;
            double t4 = 0;

            // contrast-sensitive features
            src = hist + (x + 1) * blocks[0] + (y + 1);
            for (int o = 0; o < 18; o++) {
                double h1 = min(*src * n1, 0.2);
                double h2 = min(*src * n2, 0.2);
                double h3 = min(*src * n3, 0.2);
                double h4 = min(*src * n4, 0.2);
                *dst = 0.5 * (h1 + h2 + h3 + h4);
                t1 += h1;
                t2 += h2;
                t3 += h3;
                t4 += h4;
                dst += out[0] * out[1];
                src += blocks[0] * blocks[1];
            }

            // contrast-insensitive features
            src = hist + (x + 1) * blocks[0] + (y + 1);
            for (int o = 0; o < 9; o++) {
                double sum = *src + *(src + 9 * blocks[0] * blocks[1]);
                double h1 = min(sum * n1, 0.2);
                double h2 = min(sum * n2, 0.2);
                double h3 = min(sum * n3, 0.2);
                double h4 = min(sum * n4, 0.2);
                *dst = 0.5 * (h1 + h2 + h3 + h4);
                dst += out[0] * out[1];
                src += blocks[0] * blocks[1];
            }

            // texture features
            *dst = 0.2357 * t1;
            dst += out[0] * out[1];
            *dst = 0.2357 * t2;
            dst += out[0] * out[1];
            *dst = 0.2357 * t3;
            dst += out[0] * out[1];
            *dst = 0.2357 * t4;
        }
    }
    free(hist);
    free(norm);
}

void HOG::DalalTriggsHOGdescriptorOnImage(const ImageWindowIterator &iwi,
                                          double *d_image,
                                          double *outputImage,
                                          int *windowsCenters) {
    __CLOG__
    int rowCenter, columnCenter;
    unsigned int offsetH;
    double* descriptorVector = new double[this->descriptorLengthPerWindow];
    
    // block is used by DalalTriggsHOGdescriptor
    // building this vector only once save times 
    vector<vector<vector<double> > > block(blockHeightAndWidthInCells, vector<vector<double> >
                                           (blockHeightAndWidthInCells, vector<double>
                                            (numberOfOrientationBins, 0.0) ) );
    
    // Compute all the histograms together using CUDA
    //   h_dims: dimension of one histogram
    //   numHistograms_d_h: number of histograms to compute
    //
    //       +--+--+  +--+
    // d_h = |h0|h1|..|hn|
    //       +--+--+  +--+
    //
    // where hx is a histogram
    //   and n = numHistograms_d_h
    
    const int hist1 = 2 + (this->windowHeight / this->cellHeightAndWidthInPixels);
    const int hist2 = 2 + (this->windowWidth / this->cellHeightAndWidthInPixels);
    const dim3 h_dims(hist1, hist2, this->numberOfOrientationBins);
    const unsigned int numHistograms_d_h = iwi._numberOfWindowsVertically*iwi._numberOfWindowsHorizontally;
    const unsigned long int h_size = h_dims.x * h_dims.y * h_dims.z * numHistograms_d_h;
    unsigned long long int d_h_size_t = h_size * sizeof(double);
    
    double *d_h = 0;
    double *h = new double[h_size]; // contains all the histograms
    
    const dim3 dimBlock(MAX_THREADS_2D, MAX_THREADS_2D, 1);
    const dim3 dimGrid((this->windowWidth * iwi._numberOfWindowsHorizontally + dimBlock.x -1)/dimBlock.x, (this->windowHeight * iwi._numberOfWindowsVertically + dimBlock.y -1)/dimBlock.y, 1);
    
    __START__
    cudaErrorCheck_goto(hipMalloc(&d_h, d_h_size_t));
    cudaErrorCheck_goto(hipMemset(d_h, 0., d_h_size_t));
    __STOP("@ Malloc & Memset for <d_h> @")
    
    // Compute values for histograms
    __START__
    DalalTriggsHOGdescriptor_compute_histograms<<<dimGrid, dimBlock>>>(d_h, h_dims,
                                                                       d_image, iwi._imageHeight, iwi._imageWidth,
                                                                       this->windowHeight, this->windowWidth, this->numberOfChannels,
                                                                       this->numberOfOrientationBins, this->cellHeightAndWidthInPixels,
                                                                       this->enableSignedGradients ? 1 : 0 /*signedOrUnsignedGradients*/,
                                                                       (1 + (this->enableSignedGradients ? 1 : 0)) * pi / this->numberOfOrientationBins /*binsSize*/,
                                                                       numHistograms_d_h, iwi._numberOfWindowsVertically,
                                                                       iwi._numberOfWindowsHorizontally,
                                                                       iwi._enablePadding, iwi._windowStepVertical, iwi._windowStepHorizontal);
    cudaErrorCheck_goto(hipDeviceSynchronize()); // block until the device is finished
    __STOP("@ Kernel: compute_histograms @")
    
    __START__
    cudaErrorCheck_goto(hipMemcpy(h, d_h, h_size * sizeof(double), hipMemcpyDeviceToHost));
    cudaErrorCheck_goto(hipFree(d_h));
    d_h = 0;
    __STOP("@ Memcpy & Free for <d_h> @")
    
    // Histogram normalization
    // & windowsCenters initialization
    //
    // Everything is done with native-C code (ie. without any CUDA implementation)
    __START__
    for (unsigned int windowIndexVertical = 0; windowIndexVertical < iwi._numberOfWindowsVertically; windowIndexVertical++) {
        for (unsigned int windowIndexHorizontal = 0; windowIndexHorizontal < iwi._numberOfWindowsHorizontally; windowIndexHorizontal++) {
            // Find window limits
            if (!iwi._enablePadding) {
                rowCenter = windowIndexVertical*iwi._windowStepVertical
                            + (int)round((double)iwi._windowHeight / 2.0) - 1;
                columnCenter = windowIndexHorizontal*iwi._windowStepHorizontal
                            + (int)round((double)iwi._windowWidth / 2.0) - 1;
            } else {
                rowCenter = windowIndexVertical*iwi._windowStepVertical;
                columnCenter = windowIndexHorizontal*iwi._windowStepHorizontal;
            }
            offsetH = h_dims.x * h_dims.y * h_dims.z * (windowIndexVertical + iwi._numberOfWindowsVertically * windowIndexHorizontal);

            // Compute descriptor of window
            DalalTriggsHOGdescriptor(h, offsetH, this->numberOfOrientationBins,
                             this->cellHeightAndWidthInPixels,
                             this->blockHeightAndWidthInCells,
                             this->enableSignedGradients,
                             this->l2normClipping,
                             iwi._imageHeight, iwi._imageWidth,
                             this->windowHeight, this->windowWidth,
                             this->numberOfChannels,
                             descriptorVector, block);

            // Store results
            for (unsigned int d = 0; d < this->descriptorLengthPerWindow; d++)
                outputImage[windowIndexVertical+iwi._numberOfWindowsVertically*(windowIndexHorizontal+iwi._numberOfWindowsHorizontally*d)] = descriptorVector[d];
            windowsCenters[windowIndexVertical+iwi._numberOfWindowsVertically*windowIndexHorizontal] = rowCenter;
            windowsCenters[windowIndexVertical+iwi._numberOfWindowsVertically*(windowIndexHorizontal+iwi._numberOfWindowsHorizontally)] = columnCenter;
        }
    }
    __STOP("@ Histogram Normalization @")
    
    delete[] descriptorVector;
    descriptorVector = NULL;
    delete[] h;
    h = NULL;
    return;
    
onfailure:
    hipFree(d_h);
    
    // Free temporary matrices
    delete[] descriptorVector;
    delete[] h;
    return;
}

// DALAL & TRIGGS: Histograms of Oriented Gradients for Human Detection
void DalalTriggsHOGdescriptor(double *h,
                              unsigned int offsetH,
                              unsigned int numberOfOrientationBins,
                              unsigned int cellHeightAndWidthInPixels,
                              unsigned int blockHeightAndWidthInCells,
                              bool signedOrUnsignedGradientsBool,
                              double l2normClipping,
                              unsigned int imageHeight, unsigned int imageWidth,
                              unsigned int windowHeight, unsigned int windowWidth,
                              unsigned int numberOfChannels,
                              double *descriptorVector,
                              vector<vector<vector<double> > > block) {
   
    // Variables
    //  * Compute gradients & Compute histograms
    
    const int hist1 = 2 + (windowHeight / cellHeightAndWidthInPixels);
    const int hist2 = 2 + (windowWidth / cellHeightAndWidthInPixels);
    const dim3 h_dims(hist1, hist2, numberOfOrientationBins);
    const unsigned int factor_z_dim = h_dims.x * h_dims.y;
    const unsigned int factor_y_dim = h_dims.x;
    
    //  * Block normalization
    
    int descriptorIndex(0);
     
    // Block normalization
    
    for (unsigned int x = 1; x < hist2 - blockHeightAndWidthInCells; x++) {
        for (unsigned int y = 1; y < hist1 - blockHeightAndWidthInCells; y++) {
            float blockNorm(0);
            for (unsigned int i = 0; i < blockHeightAndWidthInCells; i++)
                for (unsigned int j = 0; j < blockHeightAndWidthInCells; j++)
                    for (unsigned int k = 0; k < numberOfOrientationBins; k++)
                        blockNorm += h[offsetH + y+i + (x+j) * factor_y_dim + k * factor_z_dim] * h[offsetH + y+i + (x+j) * factor_y_dim + k * factor_z_dim];

            blockNorm = sqrt(blockNorm);
            for (unsigned int i = 0; i < blockHeightAndWidthInCells; i++) {
                for (unsigned int j = 0; j < blockHeightAndWidthInCells; j++) {
                    for (unsigned int k = 0; k < numberOfOrientationBins; k++) {
                        if (blockNorm > 0) {
                            block[i][j][k] = h[offsetH + y+i + (x+j) * factor_y_dim + k * factor_z_dim] / blockNorm;
                            if (block[i][j][k] > l2normClipping)
                                block[i][j][k] = l2normClipping;
                        } else {
                            block[i][j][k] = 0;
                        }
                    }
                }
            }

            blockNorm = 0;
            for (unsigned int i = 0; i < blockHeightAndWidthInCells; i++)
                for (unsigned int j = 0; j < blockHeightAndWidthInCells; j++)
                    for (unsigned int k = 0; k < numberOfOrientationBins; k++)
                        blockNorm += block[i][j][k] * block[i][j][k];

            blockNorm = sqrt(blockNorm);
            for (unsigned int i = 0; i < blockHeightAndWidthInCells; i++) {
                for (unsigned int j = 0; j < blockHeightAndWidthInCells; j++) {
                    for (unsigned int k = 0; k < numberOfOrientationBins; k++) {
                        if (blockNorm > 0)
                            descriptorVector[descriptorIndex] =
                                block[i][j][k] / blockNorm;
                        else
                            descriptorVector[descriptorIndex] = 0.0;
                        descriptorIndex++;
                    }
                }
            }
        }
    }
}

/* Kernels */

__device__ double atomicAdd(double* address, double val) {
    // http://stackoverflow.com/questions/16882253/cuda-atomicadd-produces-wrong-result
    unsigned long long int* address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;
    do
    {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

#define getInImage(i,j,k) ((i+rowFrom<0 || i+rowFrom>imageHeight-1 || j+columnFrom<0 || j+columnFrom>imageWidth-1) ? 0. : d_inputImage[(i+rowFrom) + imageHeight*((j+columnFrom) + imageWidth*k)])
__global__ void DalalTriggsHOGdescriptor_compute_histograms(double *d_h,
                                                               const dim3 h_dims,
                                                               const double *d_inputImage,
                                                               const unsigned int imageHeight,
                                                               const unsigned int imageWidth,
                                                               const unsigned int windowHeight,
                                                               const unsigned int windowWidth,
                                                               const unsigned int numberOfChannels,
                                                               const unsigned int numberOfOrientationBins,
                                                               const unsigned int cellHeightAndWidthInPixels,
                                                               const unsigned signedOrUnsignedGradients,
                                                               const double binsSize,
                                                               const int numHistograms,
                                                               const int numberOfWindowsVertically,
                                                               const int numberOfWindowsHorizontally,
                                                               const bool enablePadding,
                                                               const int windowStepVertical, const int windowStepHorizontal) {
    // Compute histograms values
    
    // Retrieve pixel position
    int x_ = blockIdx.x * blockDim.x + threadIdx.x;
    if (x_ >= numberOfWindowsHorizontally * windowWidth)
        return;
    int y_ = blockIdx.y * blockDim.y + threadIdx.y;
    if (y_ >= numberOfWindowsVertically * windowHeight)
        return;
    
    int x = x_ % windowWidth;
    int windowIndexHorizontal = x_ / windowWidth;
    
    int y = y_ % windowHeight;
    int windowIndexVertical = y_ / windowHeight;
    
    unsigned int factor_y_dim = h_dims.x;
    unsigned int factor_z_dim = factor_y_dim * h_dims.y;
    unsigned int factor_o_dim = factor_z_dim * h_dims.z;
    
    int offsetWindow = factor_o_dim * (windowIndexVertical + numberOfWindowsVertically * windowIndexHorizontal);
    int rowFrom, columnFrom;
    if (enablePadding) {
        rowFrom = windowIndexVertical*windowStepVertical;
        columnFrom = windowIndexHorizontal*windowStepHorizontal;
    } else {
        rowFrom = windowIndexVertical*windowStepVertical - (int)round((double)windowHeight / 2.0) + 1;
        columnFrom = windowIndexHorizontal*windowStepHorizontal - (int)ceil((double)windowWidth / 2.0) + 1;
    }
     
    // Compute deltas
    double dx[3], dy[3];
    
    if (x == 0) {
        for (unsigned int z = 0; z < numberOfChannels; z++)
            dx[z] = getInImage(y, x+1, z);
    } else {
        if (x == windowWidth - 1) {
            for (unsigned int z = 0; z < numberOfChannels; z++)
                dx[z] = -getInImage(y, x-1, z);
        } else {
            for (unsigned int z = 0; z < numberOfChannels; z++)
                dx[z] = getInImage(y, x+1, z) - getInImage(y, x-1, z);
        }
    }

    if(y == 0) {
        for (unsigned int z = 0; z < numberOfChannels; z++)
            dy[z] = -getInImage(y+1, x, z);
    } else {
        if (y == windowHeight - 1) {
            for (unsigned int z = 0; z < numberOfChannels; z++)
                dy[z] = getInImage(y-1, x, z);
        } else {
            for (unsigned int z = 0; z < numberOfChannels; z++)
                dy[z] = -getInImage(y+1, x, z) + getInImage(y-1, x, z);
        }
    }

    // Choose dominant channel based on magnitude
    double gradientMagnitude = sqrt(dx[0] * dx[0] + dy[0] * dy[0]);
    double gradientOrientation = atan2(dy[0], dx[0]);
    if (numberOfChannels > 1) {
        double tempMagnitude = gradientMagnitude;
        for (unsigned int cli = 1 ; cli < numberOfChannels ; ++cli) {
            tempMagnitude= sqrt(dx[cli] * dx[cli] + dy[cli] * dy[cli]);
            if (tempMagnitude > gradientMagnitude) {
                gradientMagnitude = tempMagnitude;
                gradientOrientation = atan2(dy[cli], dx[cli]);
            }
        }
    }

    if (gradientOrientation < 0)
        gradientOrientation += pi + (signedOrUnsignedGradients == 1) * pi;

    // Trilinear interpolation
    int bin1 = (gradientOrientation / binsSize) - 1;
    unsigned int bin2 = bin1 + 1;
    int x1   = x / cellHeightAndWidthInPixels;
    int x2   = x1 + 1;
    int y1   = y / cellHeightAndWidthInPixels;
    int y2   = y1 + 1;
    
    double Xc = (x1 + 1 - 1.5) * cellHeightAndWidthInPixels + 0.5;
    double Yc = (y1 + 1 - 1.5) * cellHeightAndWidthInPixels + 0.5;
    double Oc = (bin1 + 1 + 1 - 1.5) * binsSize;
    
    if (bin2 == numberOfOrientationBins)
        bin2 = 0;
    
    if (bin1 < 0)
        bin1 = numberOfOrientationBins - 1;
    
    // Compute histograms
    //  using reduce-pattern
    //
    // d_h needs to be set to 0.
    
    atomicAdd(
            &d_h[offsetWindow + y1 + x1*factor_y_dim + bin1*factor_z_dim],
            gradientMagnitude *
                (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                (1-((gradientOrientation-Oc)/binsSize)));
    atomicAdd(
            &d_h[offsetWindow + y1 + x1*factor_y_dim + bin2*factor_z_dim],
            gradientMagnitude *
                (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                (((gradientOrientation-Oc)/binsSize)));
    atomicAdd(
            &d_h[offsetWindow + y2 + x1*factor_y_dim + bin1*factor_z_dim],
            gradientMagnitude *
                (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                (1-((gradientOrientation-Oc)/binsSize)));
    atomicAdd(
            &d_h[offsetWindow + y2 + x1*factor_y_dim + bin2*factor_z_dim],
            gradientMagnitude *
                (1-((x+1-Xc)/cellHeightAndWidthInPixels)) *
                (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                (((gradientOrientation-Oc)/binsSize)));
    atomicAdd(
            &d_h[offsetWindow + y1 + x2*factor_y_dim + bin1*factor_z_dim],
            gradientMagnitude *
                (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                (1-((gradientOrientation-Oc)/binsSize)));
    atomicAdd(
            &d_h[offsetWindow + y1 + x2*factor_y_dim + bin2*factor_z_dim],
            gradientMagnitude *
                (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                (1-((y+1-Yc)/cellHeightAndWidthInPixels)) *
                (((gradientOrientation-Oc)/binsSize)));
    atomicAdd(
            &d_h[offsetWindow + y2 + x2*factor_y_dim + bin1*factor_z_dim],
            gradientMagnitude *
                (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                (1-((gradientOrientation-Oc)/binsSize)));
    atomicAdd(
            &d_h[offsetWindow + y2 + x2*factor_y_dim + bin2*factor_z_dim],
            gradientMagnitude *
                (((x+1-Xc)/cellHeightAndWidthInPixels)) *
                (((y+1-Yc)/cellHeightAndWidthInPixels)) *
                (((gradientOrientation-Oc)/binsSize)));
}
